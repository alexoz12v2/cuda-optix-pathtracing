#include "hip/hip_runtime.h"
#ifndef __NVCC__
    #define __NVCC__
#endif

#include "cuda-queue.h"

extern "C" __global__ void saxpy_grid_stride(int n, float a, float const* x, float* y)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < n; i += blockDim.x * gridDim.x)
    {
        y[i] = a * x[i] + y[i];
    }
}

extern "C" __global__ void kqueueDouble(dmt::ManagedQueue<int>* queue, dmt::ManagedQueue<int>* queue1)
{
    int num = 0;
    if (!queue->popDevice(&num))
        return;
    queue1->pushDevice(num * 2);
}

extern "C" __global__ void kmmqDouble(dmt::ManagedMultiQueue<double, int>* queue, dmt::ManagedMultiQueue<double, int>* queue1)
{
    int    num  = 0;
    double fnum = 0.0;
    if (!queue->popDevice(&fnum, &num))
        return;
    queue1->pushDevice(fnum * 2, num * 3);
}
