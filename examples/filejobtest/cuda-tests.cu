#include "hip/hip_runtime.h"
#include "cuda-tests.h"

#define DMT_INTERFACE_AS_HEADER
#include "dmtmacros.h"
#include "platform/platform-cuda-utils.cuh"
#include "platform/platform.h"

#include <hip/hip_runtime.h>

__global__ void fillBufferKernel(uint8_t* buffer, size_t size, uint8_t value)
{
    size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size)
        buffer[idx] = value;
}


void testBuddyDirectly(dmt::AppContext& actx, dmt::BaseMemoryResource* pMemRes)
{
    using namespace std::string_view_literals;
    dmt::BuddyMemoryResource* pBuddy = dynamic_cast<dmt::BuddyMemoryResource*>(pMemRes);
    assert(pBuddy && "incorrect allocator type");
    actx.error("THis is an error {}", {"err"sv});

    hipError_t err = hipGetLastError();
    assert(err == ::hipSuccess && "the start is already promising...");

    // tests ...
    // === Test 1: Basic Allocation ===
    {
        constexpr size_t blockSize = 256; // Assume a block size for testing
        void*            ptr1      = pBuddy->allocate(blockSize, alignof(std::max_align_t));
        assert(ptr1 != nullptr && "Allocation failed for basic block size");

        void* ptr2 = pBuddy->allocate(blockSize, alignof(std::max_align_t));
        assert(ptr2 != nullptr && ptr1 != ptr2 && "Allocation failed or returned duplicate pointer");

        pBuddy->deallocate(ptr1, blockSize, alignof(std::max_align_t));
        pBuddy->deallocate(ptr2, blockSize, alignof(std::max_align_t));
    }

    // === Test 2: Large Allocation (Edge Case) ===
    {
        size_t const largeBlockSize = pBuddy->maxBlockSize();
        void*        ptr            = pBuddy->allocate(largeBlockSize, alignof(std::max_align_t));
        assert(ptr != nullptr && "Allocation failed for large block size");

        pBuddy->deallocate(ptr, largeBlockSize, alignof(std::max_align_t));
    }

    // === Test 3: Exhaustion and Reallocation ===
    {
        std::vector<void*> allocations;
        size_t const       blockSize = 256;
        while (true)
        {
            void* ptr = pBuddy->allocate(blockSize, alignof(std::max_align_t));
            if (!ptr)
                break; // Stop allocating once the pool is exhausted
            allocations.push_back(ptr);
        }

        // Ensure all allocated memory is deallocated
        for (void* ptr : allocations)
            pBuddy->deallocate(ptr, blockSize, alignof(std::max_align_t));

        // Verify memory can be reallocated after full deallocation
        void* ptr = pBuddy->allocate(blockSize, alignof(std::max_align_t));
        assert(ptr != nullptr && "Reallocation after exhaustion failed");
        pBuddy->deallocate(ptr, blockSize, alignof(std::max_align_t));
    }

    // === Test 4: Basic CUDA Kernel Test ===
    {
        constexpr size_t  bufferSize = 1024; // Buffer size in bytes
        constexpr uint8_t fillValue  = 42;   // Value to fill the buffer with

        // Allocate device memory using the BuddyMemoryResource
        void* deviceBuffer = pBuddy->allocate(bufferSize, alignof(std::max_align_t));
        assert(deviceBuffer != nullptr && "Failed to allocate device buffer");

        // Launch the kernel to fill the buffer
        int threadsPerBlock = 256;
        int blocksPerGrid   = (bufferSize + threadsPerBlock - 1) / threadsPerBlock;
        fillBufferKernel<<<blocksPerGrid, threadsPerBlock>>>(static_cast<uint8_t*>(deviceBuffer), bufferSize, fillValue);

        // Check for kernel launch errors
        hipError_t err = hipGetLastError();
        assert(err == ::hipSuccess && "Kernel launch failed");

        // Synchronize to ensure kernel execution is complete
        err = hipDeviceSynchronize();
        assert(err == ::hipSuccess && "Device synchronization failed");

        // Allocate host memory for verification
        std::vector<uint8_t> hostBuffer(bufferSize);

        // Copy the device buffer back to the host
        err = hipMemcpy(hostBuffer.data(), deviceBuffer, bufferSize, ::hipMemcpyDeviceToHost);
        assert(err == ::hipSuccess && "Failed to copy device buffer to host");

        // Verify the buffer contents
        std::string content;
        for (size_t i = 0; i < bufferSize; ++i)
        {
            content += std::to_string(hostBuffer[i]) + ", ";
            assert(hostBuffer[i] == fillValue && "Buffer verification failed");
        }
        content.resize(content.size() - 2);

        actx.log("Buffer content: \{");
        std::string_view str       = content;
        size_t           remaining = str.size();
        size_t const     maxPrint  = actx.maxLogArgBytes() >> 1;
        size_t           offset    = 0;
        while (offset < str.size())
        {
            size_t           toPrint = std::min(remaining, actx.maxLogArgBytes());
            std::string_view s       = str.substr(offset, toPrint);
            actx.log(" {}", {s});
            offset += toPrint;
            remaining -= toPrint;
        }
        actx.log("\} End Buffer content");

        // Clean up
        pBuddy->deallocate(deviceBuffer, bufferSize, alignof(std::max_align_t));
        actx.log("CUDA Kernel Test Passed {}", {"success"sv});
    }
}
