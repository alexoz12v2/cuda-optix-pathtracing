#include "hip/hip_runtime.h"
#include "printf.cuh"

#define DMT_ENTRY_POINT
#include <platform/platform.h>

#include <glad/gl.h>
#include <GLFW/glfw3.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <array>
#include <algorithm>
#include <bit>
#include <memory>
#include <iostream>
#include <tuple>
#include <locale>
#include <codecvt>
#include <string>
#include <string_view>
#include <source_location>

#include <cstdint>
#include <cstdlib>

namespace dmt {
    template <std::floating_point T>
    struct UTF8Formatter<T>
    {
        DMT_CPU_GPU inline constexpr void operator()(T const& value, char8_t* _buffer, uint32_t& _offset, uint32_t& _bufferSize)
        {
            if (_bufferSize < _offset + 2 * sizeof(uint32_t))
                return; // Not enough space for metadata

            char* writePos = reinterpret_cast<char*>(_buffer + _offset + 2 * sizeof(uint32_t));
#if defined(__CUDA_ARCH__)
            int bytesWritten = ::dmt::snprintf(writePos,
                                               _bufferSize - _offset - 2 * sizeof(uint32_t),
                                               "%.6g",
                                               value); // Adjust precision if needed
#else
            int bytesWritten = std::snprintf(writePos, _bufferSize - _offset - 2 * sizeof(uint32_t), "%.6g", value); // Adjust precision if needed
#endif

            if (bytesWritten > 0 && static_cast<uint32_t>(bytesWritten) <= (_bufferSize - _offset - 2 * sizeof(uint32_t)))
            {
                uint32_t numBytes = static_cast<uint32_t>(bytesWritten);
                uint32_t len      = numBytes; // Each byte corresponds to one character in this case

                *std::bit_cast<uint32_t*>(_buffer + _offset)                    = numBytes; // Store `numBytes`
                *std::bit_cast<uint32_t*>(_buffer + _offset + sizeof(uint32_t)) = len;      // Store `len`

                _offset += 2 * sizeof(uint32_t) + numBytes;
                _bufferSize -= 2 * sizeof(uint32_t) + numBytes;
            }
            else
            {
                _bufferSize = 0; // Insufficient buffer space
            }
        }
    };

    template <std::integral T>
    struct UTF8Formatter<T>
    {
        DMT_CPU_GPU inline constexpr void operator()(T const& value, char8_t* _buffer, uint32_t& _offset, uint32_t& _bufferSize)
        {
            if (_bufferSize < _offset + 2 * sizeof(uint32_t))
                return; // Not enough space for metadata

            char* writePos = reinterpret_cast<char*>(_buffer + _offset + 2 * sizeof(uint32_t));
#if defined(__CUDA_ARCH__)
            int bytesWritten = ::dmt::snprintf(writePos, _bufferSize - _offset - 2 * sizeof(uint32_t), "%d", value);
#else
            int bytesWritten = std::snprintf(writePos, _bufferSize - _offset - 2 * sizeof(uint32_t), "%d", value);
#endif

            if (bytesWritten > 0 && static_cast<uint32_t>(bytesWritten) <= (_bufferSize - _offset - 2 * sizeof(uint32_t)))
            {
                uint32_t numBytes = static_cast<uint32_t>(bytesWritten);
                uint32_t len      = numBytes; // Each byte corresponds to one character in this case

                *std::bit_cast<uint32_t*>(_buffer + _offset)                    = numBytes; // Store `numBytes`
                *std::bit_cast<uint32_t*>(_buffer + _offset + sizeof(uint32_t)) = len;      // Store `len`

                _offset += 2 * sizeof(uint32_t) + numBytes;
                _bufferSize -= 2 * sizeof(uint32_t) + numBytes;
            }
            else
            {
                _bufferSize = 0; // Insufficient buffer space
            }
        }
    };

    // TODO pstd::string_view
    template <std::convertible_to<std::string_view> T>
    struct UTF8Formatter<T>
    {
        DMT_CPU_GPU inline constexpr void operator()(T const& value, char8_t* _buffer, uint32_t& _offset, uint32_t& _bufferSize)
        {
            std::string_view strView = value;

            if (_bufferSize < _offset + 2 * sizeof(uint32_t))
                return; // Not enough space for metadata

            uint32_t numBytes = static_cast<uint32_t>(strView.size());
            uint32_t len = static_cast<uint32_t>(strView.size()); // Assuming valid UTF-8 input where each character is 1 byte

            if (_bufferSize < _offset + 2 * sizeof(uint32_t) + numBytes)
                return; // Insufficient space for the string

            *std::bit_cast<uint32_t*>(_buffer + _offset)                    = numBytes; // Store `numBytes`
            *std::bit_cast<uint32_t*>(_buffer + _offset + sizeof(uint32_t)) = len;      // Store `len`

            memcpy(_buffer + _offset + 2 * sizeof(uint32_t), strView.data(), numBytes);

            _offset += 2 * sizeof(uint32_t) + numBytes;
            _bufferSize -= 2 * sizeof(uint32_t) + numBytes;
        }
    };


} // namespace dmt

using namespace dmt;

static __global__ void kPrint(char* buffer, int32_t bufferSize)
{
    int32_t gid = globalThreadIndex();
    if (gid == 0)
    {
        ::dmt::snprintf(buffer, bufferSize, "Hello From Device gid = %d\n", gid);
    }
    __syncthreads();
}

static void testDevicePrint()
{
    char*         buffer;
    int32_t const bufferSize = 4096;
    hipError_t   err        = hipMallocManaged(&buffer, bufferSize);
    assert(err == ::hipSuccess);

    kPrint<<<1, 32>>>(buffer, bufferSize);
    err = hipGetLastError();
    assert(err == ::hipSuccess);
    err = hipDeviceSynchronize();
    assert(err == ::hipSuccess);

    std::cout << buffer << std::endl;

    err = hipFree(buffer);
    assert(err == ::hipSuccess);
}

static __global__ void kContext()
{
    int32_t gid = globalThreadIndex();
    Context ctx;
    ctx.warn("fdsafdaaf\xf0\x9f\x98\x8a {}", std::make_tuple(3.f));
}

static int32_t s_keyPressed = 0;

static __global__ void kWriteFile(os::CudaFileMapping* pFileMapping, uint32_t chunkSize)
{
    pFileMapping->requestChunk(0, 0);
    uint32_t offset = threadIdx.x * (chunkSize >> 4);
    // TODO: https://docs.nvidia.com/cuda/parallel-thread-execution/#data-movement-and-conversion-instructions-st
    int* p = std::bit_cast<int*>(std::bit_cast<uintptr_t>(pFileMapping->target) + offset);
    p[0]   = threadIdx.x;
    p[1]   = threadIdx.y;
    pFileMapping->signalCompletion();
}

static void testNewContext()
{
    using namespace std::string_view_literals;
    auto res = ctx::addContext(true);
    if (res != ctx::ECtxReturn::eCreatedOnManaged)
        std::abort();
    ctx::cs->setActive(0);

    // context is available here
    {
        Context ctx;
        ctx.impl()->addHandler([](LogHandler& _out) { createConsoleHandler(_out); });
        // this is equivalent to "fdsafdaaf\xf0\x9f\x98\x8a {}", but NOT u8"fdsafdaaf\xf0\x9f\x98\x8a {}"
        //static constexpr char8_t fmtstr[] = {u8'f', u8'd', u8's', u8'a', u8'f', u8'd', u8's', u8'a', u8'f', 0xF0, 0x9F, 0x98, 0x8A, u8'{', u8'}'};
        ctx.warn("fdsafdaaf\xf0\x9f\x98\x8a {}", std::make_tuple(3.f));
        ctx.error("fdsafdaaf\xf0\x9f\x98\x8a {}", std::make_tuple(3.f));
        ctx.log("fdsafdaaf\xf0\x9f\x98\x8a {}", std::make_tuple(3.f));
        ctx.trace("fdsafdaaf\xf0\x9f\x98\x8a {}", std::make_tuple(3.f));
        ctx.flush();
        kContext<<<1, 32>>>();
        hipError_t err = hipDeviceSynchronize();
        if (err != ::hipSuccess)
            ctx.error("Failed context kernel execution \xF0\x9F\x99\x81", std::make_tuple());

        static constexpr uint32_t _64KB  = 64 * 1024;
        void*                     memory = nullptr;
        hipMalloc(&memory, _64KB); // LEAKED

        os::CudaFileMapping* pFileMapping = nullptr;

        err = hipMallocManaged(&pFileMapping, sizeof(os::CudaFileMapping));
        if (err != ::hipSuccess)
            std::abort();
        std::construct_at(pFileMapping, "bonk.txt"sv, _64KB, true, memory);
        std::atomic<int32_t> stopRequested = 2;

        ctx.log("roba {}", std::make_tuple(*pFileMapping->chunky()));

        auto const iothreadEntrypoint = [](os::CudaFileMapping* _pFileMapping, std::atomic<int32_t>* _pStopRequested) {
            while (_pStopRequested->load(std::memory_order_acquire) != 0)
                std::this_thread::yield();
            while (_pStopRequested->load(std::memory_order_acquire) != 1)
            {
                if (_pFileMapping->requestedChunk())
                {
                    // TODO once requested, use janitor
                    _pFileMapping->signalChunkLoaded();
                    _pFileMapping->waitForCompletion();
                }
            }
        };

        // TODO: The kernel crashes. fix it
        std::thread t{iothreadEntrypoint, pFileMapping, &stopRequested};
        kWriteFile<<<1, 32>>>(pFileMapping, _64KB);
        err = hipGetLastError();
        if (err != ::hipSuccess)
            __debugbreak();
        stopRequested.store(0, std::memory_order_release);

        while (!s_keyPressed) // hang
        {
        }

        stopRequested.store(1, std::memory_order_release);
        hipDeviceSynchronize();
        t.join();
        std::destroy_at(pFileMapping);
        hipFree(pFileMapping);
        hipFree(memory);
    }
}

static void keyCallback(GLFWwindow* window, int key, int scancode, int action, int mods)
{
    if (action == GLFW_PRESS)
        s_keyPressed = 1;
}

class WindowJanitor
{
public:
    WindowJanitor(GLFWkeyfun _keyCallback)
    {
        if (!glfwInit())
            std::abort();

        glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
        glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 6);
        glfwWindowHint(GLFW_OPENGL_FORWARD_COMPAT, 1);
        glfwWindowHint(GLFW_VISIBLE, GLFW_FALSE); // offscreen context
        m_window = glfwCreateWindow(640, 480, "Playground", nullptr, nullptr);
        if (!m_window)
            std::abort();
        glfwMakeContextCurrent(m_window);
        glfwSetKeyCallback(m_window, _keyCallback);
    }

    WindowJanitor(WindowJanitor const&)                = delete;
    WindowJanitor(WindowJanitor&&) noexcept            = delete;
    WindowJanitor& operator=(WindowJanitor const&)     = delete;
    WindowJanitor& operator=(WindowJanitor&&) noexcept = delete;
    ~WindowJanitor()
    {
        if (m_window)
            glfwDestroyWindow(m_window);
        glfwTerminate();
    }

private:
    GLFWwindow* m_window = nullptr;
};

int guardedMain()
{
    //std::unique_ptr<char8_t[]> ptr    = std::make_unique<char8_t[]>(2048);
    //std::unique_ptr<char8_t[]> args   = std::make_unique<char8_t[]>(2048);
    //auto                       record = createRecord(u8"afdsf {} {}",
    //                           ELogLevel::LOG,
    //                           ptr.get(),
    //                           2048,
    //                           args.get(),
    //                           2048,
    //                           std::make_tuple(3u, 3.f),
    //                           getPhysicalLocation(),
    //                           std::source_location::current());

    //std::wstring_convert<std::codecvt_utf8_utf16<wchar_t>> converter;
    //std::string                                            view{std::bit_cast<char*>(record.data), record.numBytes};
    //std::wstring                                           wstr = converter.from_bytes(view);
    //std::wcout << wstr << std::endl;

    auto v = os::getEnv();
    dmt::cudaHello(nullptr);
    WindowJanitor wj{keyCallback};

    //testDevicePrint();
    testNewContext();
    return 0;
}