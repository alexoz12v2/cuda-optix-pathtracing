#include "hip/hip_runtime.h"
#include "printf.cuh"

#define DMT_ENTRY_POINT
#include <platform/platform.h>
#include <platform/platform-cuda-utils.h>
#include <platform/platform-context.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <array>
#include <algorithm>
#include <bit>
#include <memory>
#include <iostream>
#include <tuple>
#include <locale>
#include <codecvt>
#include <string>
#include <string_view>
#include <source_location>

#include <cstdint>
#include <cstdlib>

namespace dmt {
    template <std::floating_point T>
    struct UTF8Formatter<T>
    {
        DMT_CPU_GPU constexpr inline void operator()(T const& value, char8_t* _buffer, uint32_t& _offset, uint32_t& _bufferSize)
        {
            if (_bufferSize < _offset + 2 * sizeof(uint32_t))
                return; // Not enough space for metadata

            char* writePos = reinterpret_cast<char*>(_buffer + _offset + 2 * sizeof(uint32_t));
#if defined(__CUDA_ARCH__)
            int bytesWritten = ::dmt::snprintf(writePos,
                                               _bufferSize - _offset - 2 * sizeof(uint32_t),
                                               "%.6g",
                                               value); // Adjust precision if needed
#else
            int bytesWritten = std::snprintf(writePos, _bufferSize - _offset - 2 * sizeof(uint32_t), "%.6g", value); // Adjust precision if needed
#endif

            if (bytesWritten > 0 && static_cast<uint32_t>(bytesWritten) <= (_bufferSize - _offset - 2 * sizeof(uint32_t)))
            {
                uint32_t numBytes = static_cast<uint32_t>(bytesWritten);
                uint32_t len      = numBytes; // Each byte corresponds to one character in this case

                *std::bit_cast<uint32_t*>(_buffer + _offset)                    = numBytes; // Store `numBytes`
                *std::bit_cast<uint32_t*>(_buffer + _offset + sizeof(uint32_t)) = len;      // Store `len`

                _offset += 2 * sizeof(uint32_t) + numBytes;
                _bufferSize -= 2 * sizeof(uint32_t) + numBytes;
            }
            else
            {
                _bufferSize = 0; // Insufficient buffer space
            }
        }
    };

    template <std::integral T>
    struct UTF8Formatter<T>
    {
        DMT_CPU_GPU constexpr inline void operator()(T const& value, char8_t* _buffer, uint32_t& _offset, uint32_t& _bufferSize)
        {
            if (_bufferSize < _offset + 2 * sizeof(uint32_t))
                return; // Not enough space for metadata

            char* writePos = reinterpret_cast<char*>(_buffer + _offset + 2 * sizeof(uint32_t));
#if defined(__CUDA_ARCH__)
            int bytesWritten = ::dmt::snprintf(writePos, _bufferSize - _offset - 2 * sizeof(uint32_t), "%d", value);
#else
            int bytesWritten = std::snprintf(writePos, _bufferSize - _offset - 2 * sizeof(uint32_t), "%d", value);
#endif

            if (bytesWritten > 0 && static_cast<uint32_t>(bytesWritten) <= (_bufferSize - _offset - 2 * sizeof(uint32_t)))
            {
                uint32_t numBytes = static_cast<uint32_t>(bytesWritten);
                uint32_t len      = numBytes; // Each byte corresponds to one character in this case

                *std::bit_cast<uint32_t*>(_buffer + _offset)                    = numBytes; // Store `numBytes`
                *std::bit_cast<uint32_t*>(_buffer + _offset + sizeof(uint32_t)) = len;      // Store `len`

                _offset += 2 * sizeof(uint32_t) + numBytes;
                _bufferSize -= 2 * sizeof(uint32_t) + numBytes;
            }
            else
            {
                _bufferSize = 0; // Insufficient buffer space
            }
        }
    };

    template <std::convertible_to<std::string_view> T>
    struct UTF8Formatter<T>
    {
        DMT_CPU_GPU constexpr inline void operator()(T const& value, char8_t* _buffer, uint32_t& _offset, uint32_t& _bufferSize)
        {
            std::string_view strView = value;

            if (_bufferSize < _offset + 2 * sizeof(uint32_t))
                return; // Not enough space for metadata

            uint32_t numBytes = static_cast<uint32_t>(strView.size());
            uint32_t len = static_cast<uint32_t>(strView.size()); // Assuming valid UTF-8 input where each character is 1 byte

            if (_bufferSize < _offset + 2 * sizeof(uint32_t) + numBytes)
                return; // Insufficient space for the string

            *std::bit_cast<uint32_t*>(_buffer + _offset)                    = numBytes; // Store `numBytes`
            *std::bit_cast<uint32_t*>(_buffer + _offset + sizeof(uint32_t)) = len;      // Store `len`

            memcpy(_buffer + _offset + 2 * sizeof(uint32_t), strView.data(), numBytes);

            _offset += 2 * sizeof(uint32_t) + numBytes;
            _bufferSize -= 2 * sizeof(uint32_t) + numBytes;
        }
    };


} // namespace dmt

using namespace dmt;

static __global__ void kPrint(char* buffer, int32_t bufferSize)
{
    int32_t gid = globalThreadIndex();
    if (gid == 0)
    {
        ::dmt::snprintf(buffer, bufferSize, "Hello From Device gid = %d\n", gid);
    }
    __syncthreads();
}

static void testDevicePrint()
{
    char*         buffer;
    int32_t const bufferSize = 4096;
    hipError_t   err        = hipMallocManaged(&buffer, bufferSize);
    assert(err == ::hipSuccess);

    kPrint<<<1, 32>>>(buffer, bufferSize);
    err = hipGetLastError();
    assert(err == ::hipSuccess);
    err = hipDeviceSynchronize();
    assert(err == ::hipSuccess);

    std::cout << buffer << std::endl;

    err = hipFree(buffer);
    assert(err == ::hipSuccess);
}

static void stdoutHandler(LogHandler& outLogger)
{
    outLogger.minimumLevel = ELogLevel::TRACE;
    outLogger.data         = nullptr;
    outLogger.hostFilter   = [](void* _data, LogRecord const& record) { return true; };
    outLogger.hostCallback = [](void* _data, LogRecord const& record) {
        std::wstring_convert<std::codecvt_utf8_utf16<wchar_t>> converter;
        std::string                                            view{std::bit_cast<char*>(record.data), record.numBytes};
        std::wstring                                           wstr = converter.from_bytes(view);
        std::wcout << wstr << std::endl;
    };
}

static void testNewContext()
{
    ContextImpl* impl;
    hipError_t  err = hipMallocManaged(&impl, sizeof(ContextImpl));
    assert(err == ::hipSuccess);
    std::construct_at(impl);
    stdoutHandler(*impl->addHandler());

    // context is available here
    {
        Context ctx{impl};
        ctx.log(u8"fdsafdsaf {}", std::make_tuple(3.f));
    }

    std::destroy_at(impl);
    err = hipFree(impl);
    assert(err == ::hipSuccess);
}

int guardedMain()
{
    std::unique_ptr<char8_t[]> ptr    = std::make_unique<char8_t[]>(2048);
    std::unique_ptr<char8_t[]> args   = std::make_unique<char8_t[]>(2048);
    auto                       record = createRecord(u8"afdsf {} {}",
                               ELogLevel::LOG,
                               ptr.get(),
                               2048,
                               args.get(),
                               2048,
                               std::make_tuple(3u, 3.f),
                               getPhysicalLocation(),
                               std::source_location::current());

    std::wstring_convert<std::codecvt_utf8_utf16<wchar_t>> converter;
    std::string                                            view{std::bit_cast<char*>(record.data), record.numBytes};
    std::wstring                                           wstr = converter.from_bytes(view);
    std::wcout << wstr << std::endl;

    auto v = getEnv();

    testDevicePrint();
    testNewContext();
    return 0;
}