#include "hip/hip_runtime.h"
#include "dummy.h"
// #include <cudashared/cudashared.h>

#include <cassert>

__global__ void mulKernel(float* res) {//
    // this doesn't compile. Device Linking needs to happen where device functions are consumed, but 
    // this function has already been linked to a dependency shared library, so you cannot use it
    // res[threadIdx.x] = dmt::test::multiply(2.f, 3.f);
    res[threadIdx.x] = 2.f * 3.f;
}

namespace dmt::test {

void multiplyArr(float* ptr)
{
    hipError_t cudaStatus;
    float*      d_ptr = nullptr;

    cudaStatus = hipMalloc(&d_ptr, sizeof(float) * 32);
    assert(cudaStatus == ::hipSuccess);

    mulKernel<<<1, 32>>>(d_ptr);
    assert(cudaStatus == ::hipSuccess);
    cudaStatus = hipDeviceSynchronize();
    assert(cudaStatus == ::hipSuccess);

    cudaStatus = hipMemcpy(ptr, d_ptr, 32 * sizeof(float), ::hipMemcpyDeviceToHost);
    assert(cudaStatus == ::hipSuccess);
}

}