#include "dmtmacros.h"
#include <platform/platform.h>
#include <platform/platform-cuda-utils.h>
#include <platform/platform-cuda-utils.cuh>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <algorithm>
#include <bit>
#include <limits>
#include <unordered_map>
#include <memory_resource>

#include <cstdio>
#include <cstdint>

using dmt::sid_t;
using dmt::operator""_side;

int32_t main()
{
    printf("Adding elements in the map from the host\n");
    hipFree(0); // force hipCtx_t creation

    getc(stdin);
}