#include "hip/hip_runtime.h"
#include "cudaTest.h"

#include <glad/gl.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <hip/hip_runtime.h>

#define DMT_INTERFACE_AS_HEADER
#include <platform/platform.h>


// CUDA kernel to fill the texture with gradient data
__global__ void fillTextureKernel(uchar4* devPtr, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height)
    {
        int index     = y * width + x;
        devPtr[index] = make_uchar4(x % 256, y % 256, 128, 255); // RGBA gradient
    }
}

namespace dmt
{

uint32_t createOpenGLTexture(int width, int height)
{
    GLuint texture;
    glGenTextures(1, &texture);
    glBindTexture(GL_TEXTURE_2D, texture);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA8, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);

    return texture;
}

bool RegImg(uint32_t tex, uint32_t width, uint32_t height)
{
    hipGraphicsResource_t ptrRes = nullptr;

    hipError_t reMgs = hipGraphicsGLRegisterImage(&ptrRes, tex, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone);

    if (reMgs != hipSuccess)
        return false;

    reMgs = hipGraphicsMapResources(1, &ptrRes, 0);

    if (reMgs != hipSuccess)
        return false;

    //define texture array
    hipArray_t texArray = nullptr;

    hipGraphicsSubResourceGetMappedArray(&texArray, ptrRes, 0, 0);

    // Get a device pointer to the texture memory
    uchar4* devPtr;
    size_t  pitch;

    hipMallocPitch(&devPtr, &pitch, width * sizeof(uchar4), height);

    // Launch the CUDA kernel
    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    fillTextureKernel<<<gridDim, blockDim>>>(devPtr, width, height);

    // Copy the CUDA memory to the OpenGL texture
    hipMemcpyToArray(texArray, 0, 0, devPtr, width * height * sizeof(uchar4), hipMemcpyDeviceToDevice);

    // Cleanup
    hipFree(devPtr);
    hipGraphicsUnmapResources(1, &ptrRes, 0);

    return true;
}

inline constexpr uint32_t numConstants = 2;

union SaxpyScalarConstants_Type
{
    float    f;
    uint32_t n;
};

__constant__ SaxpyScalarConstants_Type saxpyConstants[numConstants];

/**
 * Sample CUDA device function which adds an element from array A and array B.
 */
__global__ void saxpyKernel(float const* A, float const* B, float* C)
{
    uint32_t tid = blockDim.x * blockIdx.x + threadIdx.x;
    if (tid < saxpyConstants[1].n)
    {
        C[tid] = saxpyConstants[0].f * A[tid] + B[tid];
    }
}

/**
 * Wrapper function for the CUDA kernel function.
 */
void kernel(float const* A, float const* B, float scalar, float* C, uint32_t N)
{
    dmt::ConsoleLogger logger = dmt::ConsoleLogger::create();
    logger.log("Hello from a nvcc file!");

    // Launch CUDA kernel.
    float *                         d_A, *d_B, *d_C;
    SaxpyScalarConstants_Type const constants[numConstants]{{.f = scalar}, {.n = N}};

    hipMalloc((void**)&d_A, N * sizeof(float));
    hipMalloc((void**)&d_B, N * sizeof(float));
    hipMalloc((void**)&d_C, N * sizeof(float));

    hipMemcpyToSymbol(HIP_SYMBOL(saxpyConstants), constants, numConstants * sizeof(SaxpyScalarConstants_Type));

    hipMemcpy(d_A, A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * sizeof(float), hipMemcpyHostToDevice);

    dim3 const blockSize(512, 1, 1);
    dim3 const gridSize((N >> 9) + 1, 1, 1);

    saxpyKernel<<<gridSize, blockSize>>>(d_A, d_B, d_C);

    hipMemcpy(C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);
}

} // namespace dmt
