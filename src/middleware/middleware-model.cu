#include "hip/hip_runtime.h"
#define DMT_INTERFACE_AS_HEADER
#undef DMT_NEEDS_MODULE
#include "dmtmacros.h"
#include "middleware-model.h"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

namespace dmt::model {
    using namespace dmt;
    void test(AppContext& ctx) { ctx.log("AAAAAAAAAAAAAAAAAAAAAAAAAAAA"); }
} // namespace dmt::model

namespace dmt::model::soa {
    using namespace dmt;
}