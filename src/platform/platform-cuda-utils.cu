#include "hip/hip_runtime.h"
#include "platform-cuda-utils.cuh"

#define DMT_INTERFACE_AS_HEADER
#include "platform-cuda-utils.h"
#include "platform-memory.h"

#define LIBCUDACXX_ENABLE_EXPERIMENTAL_MEMORY_RESOURCE
#include <bit>

#include <cstdint>
#include <cuda/memory_resource>

namespace dmt {
    bool logCUDAStatus(MemoryContext* mctx)
    {
        if (hipError_t err = hipPeekAtLastError(); err != ::hipSuccess)
        {
            mctx->pctx.error("CUDA error: {}", {hipGetErrorString(err)});
            return false;
        }
        return true;
    }

    DMT_CPU CudaStreamHandle newStream()
    {
        hipStream_t stream;
        if (hipStreamCreate(&stream) == ::hipSuccess)
            return std::bit_cast<CudaStreamHandle>(stream);
        else
            return noStream;
    }

    DMT_CPU void deleteStream(CudaStreamHandle stream)
    {
        if (stream != noStream && stream != 0)
        {
            hipStream_t cStream = std::bit_cast<hipStream_t>(stream);
            assert(hipStreamDestroy(cStream) == ::hipSuccess);
        }
    }

    DMT_CPU CUDAHelloInfo cudaHello(MemoryContext* mctx)
    {
        CUDAHelloInfo ret;
        ret.cudaCapable = false;
        // Force cuda context lazy initialization (driver and runtime interop:
        // https://stackoverflow.com/questions/60132426/how-can-i-mix-cuda-driver-api-with-cuda-runtime-api
        int32_t count = 0;
        if (hipGetDeviceCount(&count) != ::hipSuccess || count <= 0)
        {
            mctx->pctx.error("Couldn't find any suitable CUDA capable devices in the current system");
            return ret;
        }

        int32_t        device = -1;
        hipDeviceProp_t desiredProps{};
        desiredProps.major             = 6; // minimum compute capability = 6.0
        desiredProps.canMapHostMemory  = 1;
        desiredProps.managedMemory     = 1;
        desiredProps.concurrentKernels = 1;

        if (hipChooseDevice(&device, &desiredProps) != ::hipSuccess)
        {
            mctx->pctx.error("Couldn't find any CUDA device which suits the desired requiresments");
            return ret;
        }
        ret.cudaCapable = true;
        ret.device      = device;

        hipDeviceProp_t actualProps{};
        if (hipGetDeviceProperties(&actualProps, device) != ::hipSuccess)
        {
            mctx->pctx.error("Couldn't get device {} properties", {device});
            return ret;
        }
        mctx->pctx.log("Chosed Device: {} ({})", {std::string_view{actualProps.name}, device});
        mctx->pctx.log("Compute Capability: {}.{}", {actualProps.major, actualProps.minor});
        assert(actualProps.managedMemory && actualProps.canMapHostMemory);

        ret.warpSize = actualProps.warpSize;

        // forrce CUDA context initialization (after this, you can use the CUDA driver API)
        if (hipFree(nullptr) != ::hipSuccess)
        {
            mctx->pctx.error("Couldn't initialize CUDA context");
            ret.cudaCapable = false;
            return ret;
        }
        size_t totalBytes = ret.totalMemInBytes = 0;
        if (hipMemGetInfo(nullptr, &totalBytes) != ::hipSuccess)
            mctx->pctx.error("Couldn't get the total Memory in bytes of the device");
        else
        {
            ret.totalMemInBytes = totalBytes;
            mctx->pctx.log("Total Device Memory: {}", {ret.totalMemInBytes});
        }

        if (actualProps.canMapHostMemory)
        { // all flags starts with `cudaDevice*`
            if (hipSetDeviceFlags(hipDeviceMapHost) != ::hipSuccess)
            {
                mctx->pctx.error("Failed to enable device flags for pin map host memory");
            }
        }

        return ret;
    }

    DMT_CPU void* cudaAllocate(size_t sz)
    {
        void*       tmp = nullptr;
        hipError_t err = hipMallocManaged(&tmp, sz);
        if (err != ::hipSuccess)
            return nullptr;
        return tmp;
    }

    DMT_CPU_GPU void cudaDeallocate(void* ptr, size_t sz)
    {
        if (ptr)
            hipFree(ptr);
    }

    DMT_CPU void* UnifiedMemoryResource::do_allocate(size_t _Bytes, size_t _Align) { return cudaAllocate(_Bytes); }

    DMT_CPU void UnifiedMemoryResource::do_deallocate(void* _Ptr, size_t _Bytes, size_t _Align)
    {
        cudaDeallocate(_Ptr, _Bytes);
    }

    DMT_CPU bool UnifiedMemoryResource::do_is_equal(memory_resource const& _That) const noexcept { return true; }

    void* UnifiedMemoryResource::allocateBytes(size_t sz, size_t align) { return do_allocate(sz, align); }

    void UnifiedMemoryResource::freeBytes(void* ptr, size_t sz, size_t align) { return do_deallocate(ptr, sz, align); }

    void* UnifiedMemoryResource::allocatesBytesAsync(size_t sz, size_t align, CudaStreamHandle stream)
    {
        assert(false);
        return nullptr;
    }

    void UnifiedMemoryResource::freeBytesAsync(void* ptr, size_t sz, size_t align, CudaStreamHandle stream)
    {
        assert(false);
    }

    // ----------------------------------------------------------------------------------------------------------------

    // Memory Resouce Interfaces --------------------------------------------------------------------------------------
    // cannot derive std::pmr::memory_resouce here cause we need __device__ on the allocate
    void* DeviceMemoryReosurce::allocateBytes(size_t sz, size_t align) { return do_allocate(sz, align); }
    void  DeviceMemoryReosurce::freeBytes(void* ptr, size_t sz, size_t align) { do_deallocate(ptr, sz, align); }
    void* DeviceMemoryReosurce::allocatesBytesAsync(size_t sz, size_t align, CudaStreamHandle stream)
    {
        assert(false);
        return nullptr;
    }
    void DeviceMemoryReosurce::freeBytesAsync(void* ptr, size_t sz, size_t align, CudaStreamHandle stream)
    {
        assert(false);
    }

    DMT_CPU_GPU void* DeviceMemoryReosurce::allocate(size_t sz, size_t align) { return do_allocate(sz, align); }
    DMT_CPU_GPU void  DeviceMemoryReosurce::deallocate(void* ptr, size_t sz, size_t align)
    {
        do_deallocate(ptr, sz, align);
    }
    DMT_CPU_GPU bool DeviceMemoryReosurce::operator==(DeviceMemoryReosurce const&) const noexcept { return true; }

    void* CudaAsyncMemoryReosurce::allocateBytes(size_t sz, size_t align) { return allocate(sz, align); }
    void  CudaAsyncMemoryReosurce::freeBytes(void* ptr, size_t sz, size_t align) { return deallocate(ptr, sz, align); }
    void* CudaAsyncMemoryReosurce::allocatesBytesAsync(size_t sz, size_t align, CudaStreamHandle stream)
    {
        assert(isValidHandle(stream));
        return do_allocate_async(sz, align, streamRefFromHandle(stream));
    }
    void CudaAsyncMemoryReosurce::freeBytesAsync(void* ptr, size_t sz, size_t align, CudaStreamHandle stream)
    {
        assert(isValidHandle(stream));
        do_deallocate_async(ptr, sz, align, streamRefFromHandle(stream));
    }

    DMT_CPU void* CudaAsyncMemoryReosurce::allocate_async(size_t sz, size_t align, cuda::stream_ref stream)
    {
        assert((align & (align - 1)) == 0 && "alignment should be a power of two");
        return do_allocate_async(sz, align, stream);
    }
    DMT_CPU void CudaAsyncMemoryReosurce::deallocate_async(void* ptr, size_t sz, size_t align, cuda::stream_ref stream)
    {
        assert((align & (align - 1)) == 0 && "alignment should be a power of two");
        do_deallocate_async(ptr, sz, align, stream);
    }

    // Memory Resouce Implementations ---------------------------------------------------------------------------------
    void* HostPoolReousce::allocateBytes(size_t sz, size_t align) { return m_res.allocate(sz, align); }
    void  HostPoolReousce::freeBytes(void* ptr, size_t sz, size_t align) { return m_res.deallocate(ptr, sz, align); }
    void* HostPoolReousce::allocatesBytesAsync(size_t sz, size_t align, CudaStreamHandle stream)
    {
        assert(false);
        return nullptr;
    }

    void HostPoolReousce::freeBytesAsync(void* ptr, size_t sz, size_t align, CudaStreamHandle stream) { assert(false); }

    void* HostPoolReousce::do_allocate(size_t _Bytes, size_t _Align) { return m_res.allocate(_Bytes, _Align); }
    void  HostPoolReousce::do_deallocate(void* _Ptr, size_t _Bytes, size_t _Align)
    {
        m_res.deallocate(_Ptr, _Bytes, _Align);
    }
    bool HostPoolReousce::do_is_equal(memory_resource const& _That) const noexcept { return m_res == _That; }

    DMT_CPU_GPU void* CudaMallocResource::do_allocate(size_t sz, [[maybe_unused]] size_t align)
    {
        void* tmp = nullptr;
        if (hipMalloc(&tmp, sz) != ::hipSuccess)
            return nullptr;
        return tmp;
    }

    DMT_CPU_GPU void CudaMallocResource::do_deallocate(void* ptr, size_t sz, size_t align)
    {
        assert(hipFree(ptr) == ::hipSuccess);
    }

    void* CudaMallocAsyncResource::do_allocate(size_t _Bytes, [[maybe_unused]] size_t _Align)
    {
        void* tmp = nullptr;
        if (hipMalloc(&tmp, _Bytes) != ::hipSuccess)
            return nullptr;
        return tmp;
    }
    void CudaMallocAsyncResource::do_deallocate(void* _Ptr, size_t _Bytes, [[maybe_unused]] size_t _Align)
    {
        assert(hipFree(_Ptr) == ::hipSuccess);
    }
    DMT_CPU void* CudaMallocAsyncResource::do_allocate_async(size_t sz, [[maybe_unused]] size_t align, cuda::stream_ref stream)
    {
        void* tmp = nullptr;
        if (hipMallocAsync(&tmp, sz, stream.get()) != ::hipSuccess)
            return nullptr;
        return tmp;
    }
    DMT_CPU void CudaMallocAsyncResource::do_deallocate_async(
        void*                   ptr,
        [[maybe_unused]] size_t sz,
        [[maybe_unused]] size_t align,
        cuda::stream_ref        stream)
    {
        assert(hipFreeAsync(ptr, stream.get()) == ::hipSuccess);
    }
    bool CudaMallocAsyncResource::do_is_equal(memory_resource const& _That) const noexcept { return true; }

    // Memory Resouce Boilerplate -------------------------------------------------------------------------------------
    DMT_CPU_GPU void switchOnMemoryResoure(EMemoryResourceType eAlloc, BaseMemoryResource* p, size_t* sz, bool destroy)
    {
        EMemoryResourceType category = extractCategory(eAlloc);
        EMemoryResourceType type     = extractType(eAlloc);
        switch (category)
        {
            using enum EMemoryResourceType;
            case eHost:
                switch (type)
                {
                    case ePool:
                        if (p)
                            if (destroy)
                                std::destroy_at(std::bit_cast<HostPoolReousce*>(p));
                            else
                                std::construct_at(std::bit_cast<HostPoolReousce*>(p));
                        else if (sz)
                            *sz = sizeof(HostPoolReousce);
                        break;
                }
                break;
            case eDevice:
                switch (type)
                {
                    case eCudaMalloc:
                        if (p)
                            if (destroy)
                                std::destroy_at(std::bit_cast<CudaMallocResource*>(p));
                            else
                                std::construct_at(std::bit_cast<CudaMallocResource*>(p));
                        else if (sz)
                            *sz = sizeof(CudaMallocResource);
                }
                break;
            case eAsync:
                switch (type)
                {
                    case eCudaMallocAsync:
                        if (p)
                            if (destroy)
                                std::destroy_at(std::bit_cast<CudaMallocAsyncResource*>(p));
                            else
                                std::construct_at(std::bit_cast<CudaMallocAsyncResource*>(p));
                        else if (sz)
                            *sz = sizeof(CudaMallocAsyncResource);
                }
                break;
            case eUnified:
                switch (type)
                {
                    case eCudaMallocManaged:
                        if (p)
                            if (destroy)
                                std::destroy_at(std::bit_cast<UnifiedMemoryResource*>(p));
                            else
                                std::construct_at(std::bit_cast<UnifiedMemoryResource*>(p));
                        else if (sz)
                            *sz = sizeof(UnifiedMemoryResource);
                        break;
                }
                break;
        }
    }

    DMT_CPU_GPU size_t sizeForMemoryResouce(EMemoryResourceType eAlloc)
    {
        size_t ret = 0;
        switchOnMemoryResoure(eAlloc, nullptr, &ret, true);
        return ret;
    }

    DMT_CPU_GPU BaseMemoryResource* constructMemoryResourceAt(void* ptr, EMemoryResourceType eAlloc)
    {
        BaseMemoryResource* p = std::bit_cast<BaseMemoryResource*>(ptr);
        switchOnMemoryResoure(eAlloc, p, nullptr, false);
        return p;
    }

    DMT_CPU_GPU void destroyMemoryResouceAt(BaseMemoryResource* p, EMemoryResourceType eAlloc)
    {
        switchOnMemoryResoure(eAlloc, p, nullptr, true);
    }

    DMT_CPU_GPU EMemoryResourceType categoryOf(BaseMemoryResource* allocator)
    {
        if (dynamic_cast<CudaAsyncMemoryReosurce*>(allocator))
            return EMemoryResourceType::eAsync;
        else if (dynamic_cast<DeviceMemoryReosurce*>(allocator))
            return EMemoryResourceType::eDevice;
        else if (dynamic_cast<UnifiedMemoryResource*>(allocator))
            return EMemoryResourceType::eUnified;
        else if (dynamic_cast<std::pmr::memory_resource*>(allocator))
            return EMemoryResourceType::eHost;
        assert(false);
        return EMemoryResourceType::eHost;
    }

    DMT_CPU_GPU bool isDeviceAllocator(BaseMemoryResource* allocator)
    {
        if (dynamic_cast<CudaAsyncMemoryReosurce*>(allocator))
            return true;
        else if (dynamic_cast<DeviceMemoryReosurce*>(allocator))
            return true;
        else if (dynamic_cast<UnifiedMemoryResource*>(allocator))
            return true;
        else if (dynamic_cast<std::pmr::memory_resource*>(allocator))
            return false;

        return false;
    }

    DMT_CPU_GPU bool isHostAllocator(BaseMemoryResource* allocator)
    {
        if (dynamic_cast<CudaAsyncMemoryReosurce*>(allocator))
            return false;
        else if (dynamic_cast<DeviceMemoryReosurce*>(allocator))
            return false;
        else if (dynamic_cast<UnifiedMemoryResource*>(allocator))
            return true;
        else if (dynamic_cast<std::pmr::memory_resource*>(allocator))
            return true;

        return false;
    }

    DMT_CPU_GPU void* allocateFromCategory(BaseMemoryResource* allocator, size_t sz, size_t align, CudaStreamHandle stream)
    {
        if (auto* a = dynamic_cast<CudaAsyncMemoryReosurce*>(allocator); a)
        {
            if (stream != noStream)
            {
                cuda::stream_ref streamref = streamRefFromHandle(stream);
                return a->allocate_async(sz, align, streamref);
            }
            else
                return a->allocate(sz, align);
        }
        else if (auto* a = dynamic_cast<DeviceMemoryReosurce*>(allocator); a)
            return a->allocate(sz, align);
        else if (auto* a = dynamic_cast<UnifiedMemoryResource*>(allocator); a)
            return a->allocate(sz, align);
        else if (auto* a = dynamic_cast<std::pmr::memory_resource*>(allocator); a)
            return a->allocate(sz, align);

        assert(false);
        return nullptr;
    }

    DMT_CPU_GPU void freeFromCategory(BaseMemoryResource* allocator, void* ptr, size_t sz, size_t align, CudaStreamHandle stream)
    {
        if (auto* a = dynamic_cast<CudaAsyncMemoryReosurce*>(allocator); a)
        {
            if (stream != noStream)
            {
                cuda::stream_ref streamref = streamRefFromHandle(stream);
                a->deallocate_async(ptr, sz, align, streamref);
            }
            else
                a->deallocate(ptr, sz, align);
        }
        else if (auto* a = dynamic_cast<DeviceMemoryReosurce*>(allocator); a)
            a->deallocate(ptr, sz, align);
        else if (auto* a = dynamic_cast<UnifiedMemoryResource*>(allocator); a)
            a->deallocate(ptr, sz, align);
        else if (auto* a = dynamic_cast<std::pmr::memory_resource*>(allocator); a)
            a->deallocate(ptr, sz, align);
    }

    // BaseDeviceContainer --------------------------------------------------------------------------------------------
    DMT_CPU_GPU void BaseDeviceContainer::lockForRead() const
    {
#if defined(__CUDA_ARCH__)
        // Wait until no writer is active
        while (atomicAdd(&m_writeCount, 0) > 0)
        {
            // Spin-wait
        }
        // Increment reader count
        atomicAdd(&m_readCount, 1);
#else
        std::atomic_ref<int> writeRef(m_writeCount);
        std::atomic_ref<int> readRef(m_readCount);
        // Wait until no writer is active
        while (writeRef.load(std::memory_order_acquire) > 0)
        {
            // Spin-wait
        }
        // Increment reader count
        readRef.fetch_add(1, std::memory_order_acquire);
#endif
    }

    DMT_CPU_GPU void BaseDeviceContainer::unlockForRead() const
    {
#if defined(__CUDA_ARCH__)
        // Decrement reader count
        atomicSub(&m_readCount, 1);
#else
        std::atomic_ref<int> readRef(m_readCount);
        // Decrement reader count
        readRef.fetch_sub(1, std::memory_order_release);
#endif
    }

    DMT_CPU_GPU void BaseDeviceContainer::lockForWrite() const
    {
#if defined(__CUDA_ARCH__)
        // Wait until no reader or writer is active
        while (atomicAdd(&m_readCount, 0) > 0 || atomicAdd(&m_writeCount, 0) > 0)
        {
            // Spin-wait
        }
        // Increment writer count
        atomicAdd(&m_writeCount, 1);
#else
        std::atomic_ref<int> writeRef(m_writeCount);
        std::atomic_ref<int> readRef(m_readCount);
        // Wait until no reader or writer is active
        while (readRef.load(std::memory_order_acquire) > 0 || writeRef.load(std::memory_order_acquire) > 0)
        {
            // Spin-wait
        }
        // Increment writer count
        writeRef.fetch_add(1, std::memory_order_acquire);
#endif
    }

    DMT_CPU_GPU void BaseDeviceContainer::unlockForWrite() const
    {
#if defined(__CUDA_ARCH__)
        // Decrement writer count
        atomicSub(&m_writeCount, 1);
#else
        std::atomic_ref<int> writeRef(m_writeCount);
        // Decrement writer count
        writeRef.fetch_sub(1, std::memory_order_release);
#endif
    }

    DMT_CPU_GPU void BaseDeviceContainer::waitWriter() const
    {
#if defined(__CUDA_ARCH__)
        // Spin-wait until no writer is active
        while (atomicAdd(&m_writeCount, 0) > 0)
        {
            // Spin-wait
        }
#else
        std::atomic_ref<int> writeRef(m_writeCount);
        // Spin-wait until no writer is active
        while (writeRef.load(std::memory_order_acquire) > 0)
        {
            // Spin-wait
        }
#endif
    }

    // DynaArray ------------------------------------------------------------------------------------------------------
    DMT_CPU_GPU DynaArray::DynaArray(DynaArray const& other) :
    BaseDeviceContainer(other.m_resource, other.stream),
    m_elemSize(other.m_elemSize)
    {
        other.lockForRead();
        m_resource = other.m_resource;
        stream     = other.stream;
        m_elemSize = other.m_elemSize;
        reserve(other.m_size);
        lockForWrite();
        copyFrom(other);
        unlockForWrite();
        other.unlockForRead();
    }

    DMT_CPU_GPU DynaArray::DynaArray(DynaArray&& other) noexcept : BaseDeviceContainer(other.m_resource, other.stream)
    {
        other.lockForWrite();
        m_resource = std::exchange(other.m_resource, nullptr);
        stream     = other.stream;
        m_head     = std::exchange(other.m_head, nullptr);
        m_size     = std::exchange(other.m_size, 0);
        m_capacity = std::exchange(other.m_capacity, 0);
        m_elemSize = other.m_elemSize;
        other.unlockForWrite();
    }

    DMT_CPU_GPU void DynaArray::reserve(size_t newCapacity, bool lock)
    {
        if (lock)
            lockForWrite();
        if (newCapacity <= m_capacity)
            return;

        void* newHead = allocateFromCategory(m_resource, newCapacity * m_elemSize, alignof(std::max_align_t), stream);
        if (m_head)
        {
            std::memcpy(newHead, m_head, m_size * m_elemSize);
            freeFromCategory(m_resource, m_head, m_size * m_elemSize, alignof(std::max_align_t), stream);
        }
        m_head     = newHead;
        m_capacity = newCapacity;
        if (lock)
            unlockForWrite();
    }

    DMT_CPU_GPU void DynaArray::clear(bool lock) noexcept
    {
        if (lock)
            lockForWrite();
        if (m_head)
        {
            freeFromCategory(m_resource, m_head, m_size * m_elemSize, alignof(std::max_align_t), stream);
            m_head = nullptr;
        }
        if (lock)
            unlockForWrite();
    }

    DMT_CPU_GPU bool DynaArray::push_back(void const* pValue, bool srcHost, bool lock)
    {
        bool ret = true;
        if (lock)
            lockForWrite();

        if (m_size >= m_capacity)
            reserve(m_capacity > 0 ? m_capacity >> 1 : 1, false);

        void* dest = std::bit_cast<void*>(std::bit_cast<uintptr_t>(m_head) + m_size * m_elemSize);

#if defined(__CUDA_ARCH__)
        if (srcHost) // error!
            ret = false;
        else
            std::memcpy(dest, pValue, m_elemSize);
#else
        hipMemcpyKind kind = isDeviceAllocator(m_resource)
                                  ? (srcHost ? ::hipMemcpyHostToDevice : ::hipMemcpyDeviceToDevice)
                                  : (srcHost ? ::hipMemcpyHostToHost : ::hipMemcpyDeviceToHost);
        hipError_t    res  = hipMemcpy(dest, pValue, m_elemSize, kind);
        if (res != ::hipSuccess)
            ret = false;
#endif
        if (ret)
            ++m_size;

        if (lock)
            unlockForWrite();
        return ret;
    }

    DMT_CPU_GPU void DynaArray::pop_back(bool lock)
    {
        if (lock)
            lockForWrite();

        if (m_size != 0)
            --m_size;

        if (lock)
            unlockForWrite();
    }

    // assumes you already locked for read

    DMT_CPU_GPU void const* DynaArray::at(size_t index) const
    {
        assert(index < m_size);
#if defined(__CUDA_ARCH__)
        if (isDeviceAllocator(m_resource))
            return std::bit_cast<void*>(std::bit_cast<uintptr_t>(m_head) + index * m_elemSize);
        else
            return nullptr;
#else
        if (isHostAllocator(m_resource))
            return std::bit_cast<void*>(std::bit_cast<uintptr_t>(m_head) + index * m_elemSize);
        else
        {
            assert(false);
            return nullptr;
        }
#endif
    }

    DMT_CPU_GPU void DynaArray::copyFrom(DynaArray const& other)
    {
        if (other.m_size > 0)
        {
            std::memcpy(m_head, other.m_head, other.m_size * other.m_elemSize);
            m_size = other.m_size;
        }
    }

    DMT_CPU_GPU DynaArray& DynaArray::operator=(DynaArray const& other)
    {
        if (this != &other)
        {
            lockForWrite();
            other.lockForWrite();
            assert(m_elemSize == other.m_elemSize);

            clear();
            reserve(other.m_size, false);
            copyFrom(other);

            other.unlockForWrite();
            unlockForWrite();
        }
        return *this;
    }

    DMT_CPU_GPU DynaArray& DynaArray::operator=(DynaArray&& other) noexcept
    {
        if (this != &other)
        {
            lockForWrite();
            other.lockForWrite();
            assert(m_elemSize == other.m_elemSize);

            m_size     = std::exchange(other.m_size, 0);
            m_capacity = std::exchange(other.m_capacity, 0);
            m_head     = std::exchange(other.m_head, nullptr);

            other.unlockForWrite();
            unlockForWrite();
        }
        return *this;
    }

    DMT_CPU_GPU DynaArray::~DynaArray() noexcept { clear(); }

    DMT_CPU bool DynaArray::copyToHostSync(void* /*DMT_RESTRICT*/ dest, bool lock) const
    {
        if (!isDeviceAllocator(m_resource))
        {
            assert(false);
            return false;
        }

        bool ret = true;
        if (lock)
            lockForRead();

        ret = ::hipSuccess == hipMemcpy(dest, m_head, m_size * m_elemSize, ::hipMemcpyDeviceToHost);

        if (lock)
            unlockForRead();

        return ret;
    }

    DMT_CPU_GPU size_t DynaArray::size(bool lock) const
    {
        size_t ret = 0;
        if (lock)
            lockForRead();
        ret = m_size;
        if (lock)
            unlockForRead();
        return ret;
    }
} // namespace dmt
