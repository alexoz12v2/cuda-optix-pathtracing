#include "hip/hip_runtime.h"
#define DMT_INTERFACE_AS_HEADER
#include "platform-cuda-utils.h"
#include "platform-memory.h"

#define LIBCUDACXX_ENABLE_EXPERIMENTAL_MEMORY_RESOURCE
#include <cstdint>
#include <cuda/memory_resource>

namespace dmt {
    bool logCUDAStatus(MemoryContext* mctx)
    {
        if (hipError_t err = hipPeekAtLastError(); err != ::hipSuccess)
        {
            mctx->pctx.error("CUDA error: {}", {hipGetErrorString(err)});
            return false;
        }
        return true;
    }

    DMT_CPU CUDAHelloInfo cudaHello(MemoryContext* mctx)
    {
        CUDAHelloInfo ret;
        ret.cudaCapable = false;
        // Force cuda context lazy initialization (driver and runtime interop:
        // https://stackoverflow.com/questions/60132426/how-can-i-mix-cuda-driver-api-with-cuda-runtime-api
        int32_t count = 0;
        if (hipGetDeviceCount(&count) != ::hipSuccess || count <= 0)
        {
            mctx->pctx.error("Couldn't find any suitable CUDA capable devices in the current system");
            return ret;
        }

        int32_t        device = -1;
        hipDeviceProp_t desiredProps{};
        desiredProps.major             = 6; // minimum compute capability = 6.0
        desiredProps.canMapHostMemory  = 1;
        desiredProps.managedMemory     = 1;
        desiredProps.concurrentKernels = 1;

        if (hipChooseDevice(&device, &desiredProps) != ::hipSuccess)
        {
            mctx->pctx.error("Couldn't find any CUDA device which suits the desired requiresments");
            return ret;
        }
        ret.cudaCapable = true;
        ret.device      = device;

        hipDeviceProp_t actualProps{};
        if (hipGetDeviceProperties(&actualProps, device) != ::hipSuccess)
        {
            mctx->pctx.error("Couldn't get device {} properties", {device});
            return ret;
        }
        mctx->pctx.log("Chosed Device: {}", {actualProps.name});
        mctx->pctx.log("Compute Capability: {}.{}", {actualProps.major, actualProps.minor});
        assert(actualProps.managedMemory && actualProps.canMapHostMemory);

        ret.warpSize = actualProps.warpSize;

        // forrce CUDA context initialization
        if (hipFree(nullptr) != ::hipSuccess)
        {
            mctx->pctx.error("Couldn't initialize CUDA context");
            ret.cudaCapable = false;
            return ret;
        }

        if (actualProps.canMapHostMemory)
        { // all flags starts with `cudaDevice*`
            if (hipSetDeviceFlags(hipDeviceMapHost) != ::hipSuccess)
            {
                mctx->pctx.error("Failed to enable device flags for pin map host memory");
            }
        }

        return ret;
    }

    DMT_CPU void* cudaAllocate(size_t sz)
    {
        void*       tmp = nullptr;
        hipError_t err = hipMallocManaged(&tmp, sz);
        if (err != ::hipSuccess)
            return nullptr;
        return tmp;
    }

    DMT_CPU_GPU void cudaDeallocate(void* ptr, size_t sz)
    {
        if (ptr)
            hipFree(ptr);
    }

    template <class MemoryResource>
        requires cuda::mr::resource<MemoryResource>
    void* maybe_allocate_async(MemoryResource& resource, std::size_t size, std::size_t align, cuda::stream_ref stream)
    {
        if constexpr (cuda::mr::async_resource<MemoryResource>)
        {
            return resource.allocate_async(size, align, stream);
        }
        else
        {
            return resource.allocate(size, align);
        }
    }

    DMT_CPU void* UnifiedMemoryResource::do_allocate(size_t _Bytes, size_t _Align) { return cudaAllocate(_Bytes); }

    DMT_CPU void UnifiedMemoryResource::do_deallocate(void* _Ptr, size_t _Bytes, size_t _Align)
    {
        cudaDeallocate(_Ptr, _Bytes);
    }

    DMT_CPU bool UnifiedMemoryResource::do_is_equal(memory_resource const& _That) const noexcept { return true; }
} // namespace dmt