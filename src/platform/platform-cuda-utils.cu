#include "hip/hip_runtime.h"
#define DMT_INTERFACE_AS_HEADER
#undef DMT_NEEDS_MODULE
// CANNOT USE os-utils
#include "platform-cuda-utils.cuh"
#include "platform-cuda-utils.h"
#include "platform-memory.h"

#define LIBCUDACXX_ENABLE_EXPERIMENTAL_MEMORY_RESOURCE
#include <bit>

#include <cstdint>
#include <hip/hip_runtime.h>
#include <cuda/memory_resource>

namespace dmt {
    bool logCUDAStatus(MemoryContext* mctx)
    {
        if (hipError_t err = hipPeekAtLastError(); err != ::hipSuccess)
        {
            mctx->pctx.error("CUDA error: {}", {hipGetErrorString(err)});
            return false;
        }
        return true;
    }

    __host__ CudaStreamHandle newStream()
    {
        hipStream_t stream;
        if (hipStreamCreate(&stream) == ::hipSuccess)
            return std::bit_cast<CudaStreamHandle>(stream);
        else
            return noStream;
    }

    __host__ hipDevice_t currentDeviceHandle()
    {
        hipDevice_t    ret;
        int32_t     deviceId = 0;
        hipError_t err      = hipGetDevice(&deviceId);
        assert(err == ::hipSuccess);
        hipError_t err1 = hipDeviceGet(&ret, deviceId);
        assert(err1 == ::hipSuccess);
        return ret;
    }

    __host__ void deleteStream(CudaStreamHandle stream)
    {
        if (stream != noStream && stream != 0)
        {
            hipStream_t cStream = std::bit_cast<hipStream_t>(stream);
            hipError_t  err     = hipStreamDestroy(cStream);
            assert(err == ::hipSuccess);
        }
    }

    __host__ CUDAHelloInfo cudaHello(MemoryContext* mctx)
    {
        CUDAHelloInfo ret;
        ret.cudaCapable = false;
        // Force cuda context lazy initialization (driver and runtime interop:
        // https://stackoverflow.com/questions/60132426/how-can-i-mix-cuda-driver-api-with-cuda-runtime-api
        int32_t count = 0;
        if (hipGetDeviceCount(&count) != ::hipSuccess || count <= 0)
        {
            if (mctx)
                mctx->pctx.error("Couldn't find any suitable CUDA capable devices in the current system");
            return ret;
        }

        int32_t        device = -1;
        hipDeviceProp_t desiredProps{};
        desiredProps.major             = 6; // minimum compute capability = 6.0
        desiredProps.canMapHostMemory  = 1;
        desiredProps.managedMemory     = 1;
        desiredProps.concurrentKernels = 1;

        if (hipChooseDevice(&device, &desiredProps) != ::hipSuccess)
        {
            if (mctx)
                mctx->pctx.error("Couldn't find any CUDA device which suits the desired requiresments");
            return ret;
        }
        ret.cudaCapable = true;
        ret.device      = device;

        hipDeviceProp_t actualProps{};
        if (hipGetDeviceProperties(&actualProps, device) != ::hipSuccess)
        {
            if (mctx)
                mctx->pctx.error("Couldn't get device {} properties", {device});
            return ret;
        }
        if (mctx)
        {
            mctx->pctx.log("Chosed Device: {} ({})", {std::string_view{actualProps.name}, device});
            mctx->pctx.log("Compute Capability: {}.{}", {actualProps.major, actualProps.minor});
        }
        assert(actualProps.managedMemory && actualProps.canMapHostMemory);

        ret.warpSize = actualProps.warpSize;

        // forrce CUDA context initialization (after this, you can use the CUDA driver API)
        // the context, if needed, can be fetched with `hipCtxGetCurrent`
        if (hipFree(nullptr) != ::hipSuccess)
        {
            if (mctx)
                mctx->pctx.error("Couldn't initialize CUDA context");
            ret.cudaCapable = false;
            return ret;
        }
        size_t totalBytes = ret.totalMemInBytes = 0;
        if (hipMemGetInfo(nullptr, &totalBytes) != ::hipSuccess)
            if (mctx)
                mctx->pctx.error("Couldn't get the total Memory in bytes of the device");
            else
            {
                ret.totalMemInBytes = totalBytes;
                if (mctx)
                    mctx->pctx.log("Total Device Memory: {}", {ret.totalMemInBytes});
            }

        if (actualProps.canMapHostMemory)
        { // all flags starts with `cudaDevice*`
            if (hipSetDeviceFlags(hipDeviceMapHost) != ::hipSuccess)
            {
                if (mctx)
                    mctx->pctx.error("Failed to enable device flags for pin map host memory");
            }
        }

        // check current device support for for CU_DEVICE_ATTRIBUTE_MEMORY_POOLS and CU_DEVICE_ATTRIBUTE_VIRTUAL_MEMORY
        hipDevice_t deviceHandle;
        hipError_t res;
        res = hipDeviceGet(&deviceHandle, device);
        assert(res == ::hipSuccess);
        int32_t support = 0;
        res             = hipDeviceGetAttribute(&support, hipDeviceAttributeMemoryPoolsSupported, deviceHandle);
        assert(res == ::hipSuccess);
        assert(support <= 1);
        ret.supportsMemoryPools = support;

        // support for `hipMemAddressReserve`, `hipMemCreate`, `hipMemMap` and related
        res = hipDeviceGetAttribute(&support, hipDeviceAttributeVirtualMemoryManagementSupported, deviceHandle);
        assert(res == ::hipSuccess);
        assert(support <= 1);
        ret.supportsVirtualMemory = support;

        // various inofration
        res = hipDeviceGetAttribute(&support, hipDeviceAttributeMaxBlocksPerMultiprocessor, deviceHandle);
        assert(res == ::hipSuccess);
        ret.perMultiprocessorMaxBlocks = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeMaxRegistersPerMultiprocessor, deviceHandle);
        assert(res == ::hipSuccess);
        ret.perMultiprocessor.maxRegisters = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeMaxSharedMemoryPerMultiprocessor, deviceHandle);
        assert(res == ::hipSuccess);
        ret.perMultiprocessor.maxSharedMemory = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeL2CacheSize, deviceHandle);
        assert(res == ::hipSuccess);
        ret.L2CacheBytes = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeMultiprocessorCount, deviceHandle);
        assert(res == ::hipSuccess);
        ret.multiprocessorCount = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeMaxRegistersPerBlock, deviceHandle);
        assert(res == ::hipSuccess);
        ret.perBlock.maxRegisters = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeMaxSharedMemoryPerBlock, deviceHandle);
        assert(res == ::hipSuccess);
        ret.perBlock.maxSharedMemory = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeTotalConstantMemory, deviceHandle);
        assert(res == ::hipSuccess);
        ret.constantMemoryBytes = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeMaxThreadsPerBlock, deviceHandle);
        assert(res == ::hipSuccess);
        ret.perBlockMaxThreads = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeMaxBlockDimX, deviceHandle);
        assert(res == ::hipSuccess);
        ret.maxBlockDim.x = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeMaxBlockDimY, deviceHandle);
        assert(res == ::hipSuccess);
        ret.maxBlockDim.y = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeMaxBlockDimZ, deviceHandle);
        assert(res == ::hipSuccess);
        ret.maxBlockDim.z = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeMaxGridDimX, deviceHandle);
        assert(res == ::hipSuccess);
        ret.maxGridDim.x = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeMaxGridDimY, deviceHandle);
        assert(res == ::hipSuccess);
        ret.maxGridDim.y = support;

        res = hipDeviceGetAttribute(&support, hipDeviceAttributeMaxGridDimZ, deviceHandle);
        assert(res == ::hipSuccess);
        ret.maxGridDim.z = support;

        // More details about interop between driver and runtime: https://docs.nvidia.com/cuda/cuda-runtime-api/group__CUDART__DRIVER.html
        // what concerns us is that
        // `CUStream` and `hipStream_t`
        // `hipEvent_t`  and `hipEvent_t`
        // `hipArray_t`  and `hipArray_t`
        // `hipGraphicsResource_t` and `hipGraphicsResource_t`
        // `hipTextureObject_t` and `hipTextureObject_t`
        // `CUSurfObject` and `hipSurfaceObject_t`
        // `hipFunction_t` and `hipFunction_t`
        // are *all interchangeable* by `static_cast`

        return ret;
    }

    __device__ int32_t globalThreadIndex()
    {
        int32_t const column          = threadIdx.x;
        int32_t const row             = threadIdx.y;
        int32_t const aisle           = threadIdx.z;
        int32_t const threads_per_row = blockDim.x;                  //# threads in x direction aka row
        int32_t const threads_per_aisle = (blockDim.x * blockDim.y); //# threads in x and y direction for total threads per aisle

        int32_t const threads_per_block = (blockDim.x * blockDim.y * blockDim.z);
        int32_t const rowOffset         = (row * threads_per_row);     //how many rows to push out offset by
        int32_t const aisleOffset       = (aisle * threads_per_aisle); // how many aisles to push out offset by

        //S32_t constecond section locates and caculates block offset withing the grid
        int32_t const blockColumn    = blockIdx.x;
        int32_t const blockRow       = blockIdx.y;
        int32_t const blockAisle     = blockIdx.z;
        int32_t const blocks_per_row = gridDim.x;                 //# blocks in x direction aka blocks per row
        int32_t const blocks_per_aisle = (gridDim.x * gridDim.y); // # blocks in x and y direction for total blocks per aisle
        int32_t const blockRowOffset   = (blockRow * blocks_per_row);     // how many rows to push out block offset by
        int32_t const blockAisleOffset = (blockAisle * blocks_per_aisle); // how many aisles to push out block offset by
        int32_t const blockId          = blockColumn + blockRowOffset + blockAisleOffset;

        int32_t const blockOffset = (blockId * threads_per_block);

        int32_t const gid = (blockOffset + aisleOffset + rowOffset + column);
        return gid;
    }

    __device__ int32_t warpWideThreadIndex()
    {
        int32_t id = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
        return id % warpSize;
    }

    __host__ void* cudaAllocate(size_t sz)
    {
        void*       tmp = nullptr;
        hipError_t err = hipMallocManaged(&tmp, sz);
        if (err != ::hipSuccess)
            return nullptr;
        return tmp;
    }

    __host__ __device__ void cudaDeallocate(void* ptr, size_t sz)
    {
        if (ptr)
            hipFree(ptr);
    }

    __host__ UnifiedMemoryResource::UnifiedMemoryResource() :
        BaseMemoryResource(makeMemResId(EMemoryResourceType::eUnified, EMemoryResourceType::eCudaMallocManaged))
    {
        m_host.allocateBytes = UnifiedMemoryResource::allocateBytes;
        m_host.freeBytes = UnifiedMemoryResource::freeBytes;
        m_host.allocateBytesAsync = UnifiedMemoryResource::allocateBytesAsync;
        m_host.freeBytesAsync = UnifiedMemoryResource::freeBytesAsync;
        m_host.deviceHasAccess = UnifiedMemoryResource::deviceHasAccess;
        m_host.hostHasAccess = UnifiedMemoryResource::hostHasAccess;
        initTable<<<1, 32 >>>(*this);
        hipDeviceSynchronize();
    }

    __host__ void* UnifiedMemoryResource::do_allocate(size_t sz, size_t align) {
        return UnifiedMemoryResource::allocateBytes(this, sz, align);
    }

    __host__ void UnifiedMemoryResource::do_deallocate(void* _Ptr, size_t _Bytes, size_t _Align)
    {
        UnifiedMemoryResource::freeBytes(this, _Ptr, _Bytes, _Align);
    }

    __host__ bool UnifiedMemoryResource::do_is_equal(memory_resource const& _That) const noexcept { return true; }

    __host__ __device__ void* UnifiedMemoryResource::allocateBytes(BaseMemoryResource* pAlloc, size_t sz, size_t align)
    {
#if defined(__CUDA_ARCH__)
        // you shouldn't be here
        return nullptr;
#else
        return cudaAllocate(sz);
#endif
    }

    __host__ __device__ void UnifiedMemoryResource::freeBytes(BaseMemoryResource* pAlloc, void* ptr, size_t sz, size_t align)
    {
#if !defined(__CUDA_ARCH__)
        cudaDeallocate(ptr, sz);
#endif
    }

    __host__ void* UnifiedMemoryResource::allocateBytesAsync(BaseMemoryResource* pAlloc, size_t sz, size_t align, CudaStreamHandle stream)
    {
        assert(false);
        return nullptr;
    }

    __host__ void UnifiedMemoryResource::freeBytesAsync(BaseMemoryResource* pAlloc, void* ptr, size_t sz, size_t align, CudaStreamHandle stream)
    {
        assert(false);
    }

    __host__ __device__ bool UnifiedMemoryResource::deviceHasAccess(BaseMemoryResource const* pAlloc, int32_t deviceID)
    {
        int32_t     currDev;
        hipError_t cudaErr = hipGetDevice(&currDev);
        assert(cudaErr == ::hipSuccess);
        int32_t supported = 0;
        cudaErr           = hipDeviceGetAttribute(&supported, ::hipDeviceAttributeManagedMemory, currDev);
        assert(cudaErr == ::hipSuccess);
        return supported != 0;
    }

    __host__ __device__ bool UnifiedMemoryResource::hostHasAccess(BaseMemoryResource const* pAlloc) { return true; }

    // Memory Resouce Interfaces --------------------------------------------------------------------------------------
    __host__ __device__ void* BaseMemoryResource::tryAllocateAsync(size_t sz, size_t align, CudaStreamHandle stream)
    {
#if defined(__CUDA_ARCH__)
        return allocateBytes(sz, align);
#else
        if (stream != noStream && categoryOf(this) == EMemoryResourceType::eAsync)
            return allocateBytesAsync(sz, align, stream);
        else
            return allocateBytes(sz, align);
#endif
    }

    __host__ __device__ void BaseMemoryResource::tryFreeAsync(void* ptr, size_t sz, size_t align, CudaStreamHandle stream)
    {
#if defined(__CUDA_ARCH__)
        return freeBytes(ptr, sz, align);
#else
        if (stream != noStream && categoryOf(this) == EMemoryResourceType::eAsync)
            return freeBytesAsync(ptr, sz, align, stream);
        else
            return freeBytes(ptr, sz, align);
#endif
    }

    __host__ __device__ void* BaseMemoryResource::allocateBytes(size_t sz, size_t align) {
#if defined(__CUDA_ARCH__)
        assert(m_device.allocateBytes);
        return m_device.allocateBytes(this, sz, align);
#else
        assert(m_host.allocateBytes);
        return m_host.allocateBytes(this, sz, align);
#endif
    }

    __host__ __device__ void BaseMemoryResource::freeBytes(void* ptr, size_t sz, size_t align) {
#if defined(__CUDA_ARCH__)
        assert(m_device.freeBytes);
        m_device.freeBytes(this, ptr, sz, align);
#else
        assert(m_host.freeBytes);
        m_host.freeBytes(this, ptr, sz, align);
#endif
    }

    __host__ void* BaseMemoryResource::allocateBytesAsync(size_t sz, size_t align, CudaStreamHandle stream)
    {
        assert(m_host.allocateBytesAsync);
        return m_host.allocateBytesAsync(this, sz, align, stream);
    }

    __host__ void BaseMemoryResource::freeBytesAsync(void* ptr, size_t sz, size_t align, CudaStreamHandle stream) {
        assert(m_host.freeBytesAsync);
        m_host.freeBytesAsync(this, ptr, sz, align, stream);
    }

    __host__ __device__ bool BaseMemoryResource::deviceHasAccess(int32_t deviceID) const {
#if defined(__CUDA_ARCH__)
        assert(m_device.deviceHasAccess);
        return m_device.deviceHasAccess(this, deviceID);
#else
        assert(m_host.deviceHasAccess);
        return m_host.deviceHasAccess(this, deviceID);
#endif
    }

    __host__ __device__ bool BaseMemoryResource::hostHasAccess() const {
#if defined(__CUDA_ARCH__)
        assert(m_device.hostHasAccess);
        return m_device.hostHasAccess(this);
#else
        assert(m_host.hostHasAccess);
        return m_host.hostHasAccess(this);
#endif
    }

    // Memory Allocation Helpers --------------------------------------------------------------------------------------
    size_t getDeviceAllocationGranularity(int32_t deviceId, hipMemAllocationProp* outProp)
    {
        hipMemAllocationProp prop = {};
        prop.type                = ::hipMemAllocationTypePinned;
        prop.location.type       = ::hipMemLocationTypeDevice; // allocate memory on device whose id is given
        prop.location.id         = deviceId;
        size_t   granularity     = 0;
        hipError_t res = hipMemGetAllocationGranularity(&granularity, &prop, ::hipMemAllocationGranularityMinimum);
        assert(res == ::hipSuccess);
        if (outProp)
            *outProp = prop;
        return granularity;
    }

    bool allocateDevicePhysicalMemory(int32_t deviceId, size_t size, hipMemGenericAllocationHandle_t& out)
    {
        hipMemAllocationProp prop        = {};
        size_t              granularity = getDeviceAllocationGranularity(deviceId, &prop);
        size                            = roundUpToNextMultipleOf(size, granularity);
        hipMemGenericAllocationHandle_t handle;
        auto                         result = hipMemCreate(&handle, size, &prop, 0);
        if (result == ::hipSuccess)
        {
            out = handle;
            return true;
        }
        else if (result == ::hipErrorOutOfMemory)
            return false;
        else
        {
            assert(false);
            return false;
        }
    }

    void setReadWriteDeviceVirtMemory(int32_t deviceId, hipDeviceptr_t ptr, size_t size)
    {
        hipMemAccessDesc accessDesc = {};
        accessDesc.location.type   = ::hipMemLocationTypeDevice;
        accessDesc.location.id     = deviceId;
        accessDesc.flags           = ::hipMemAccessFlagsProtReadWrite;

        hipError_t res = hipMemSetAccess(ptr, size, &accessDesc, 1);
        assert(res == ::hipSuccess);
    }

    // Memory Resouce Implementations ---------------------------------------------------------------------------------
    __host__ __device__ void* CudaMallocResource::allocateBytes(BaseMemoryResource* pAlloc, size_t sz, size_t align) {
        void* tmp = nullptr;
        if (hipMalloc(&tmp, sz) != ::hipSuccess)
            return nullptr;
        return tmp;
    }
    __host__ __device__ void  CudaMallocResource::freeBytes(BaseMemoryResource* pAlloc, void* ptr, size_t sz, size_t align) {
        hipError_t err = hipFree(ptr);
        assert(err == ::hipSuccess);
    }
    __host__ void* CudaMallocResource::allocateBytesAsync(BaseMemoryResource* pAlloc, size_t sz, size_t align, CudaStreamHandle stream) {
        assert(false);
        return nullptr;
    }
    __host__ void CudaMallocResource::freeBytesAsync(BaseMemoryResource* pAlloc, void* ptr, size_t sz, size_t align, CudaStreamHandle stream) {
        assert(false);
    }

    __host__ CudaMallocResource::CudaMallocResource() : DeviceMemoryReosurce(EMemoryResourceType::eCudaMalloc) {
        m_host.allocateBytes = CudaMallocResource::allocateBytes;
        m_host.freeBytes = CudaMallocResource::freeBytes;
        m_host.allocateBytesAsync = CudaMallocResource::allocateBytesAsync;
        m_host.freeBytesAsync = CudaMallocResource::freeBytesAsync;
        m_host.deviceHasAccess = CudaMallocResource::deviceHasAccess;
        m_host.hostHasAccess = CudaMallocResource::hostHasAccess;
        initTable<<<1, 32>>>(*this);
        hipDeviceSynchronize();
    }
    __host__ __device__ void* CudaMallocResource::allocate(size_t sz, size_t align)
    {
        return CudaMallocResource::allocateBytes(this, sz, align);
    }
    __host__ __device__ void CudaMallocResource::deallocate(void* ptr, size_t sz, size_t align)
    {
        CudaMallocResource::freeBytes(this, ptr, sz, align);
    }


    __host__ __device__ void* CudaMallocAsyncResource::allocateBytes(BaseMemoryResource* pAlloc, size_t sz, size_t align) {
#if !defined(__CUDA_ARCH__)
        return reinterpret_cast<CudaMallocAsyncResource*>(pAlloc)->allocate(sz, align);
#else
        assert(false);
        return nullptr;
#endif
    }
    __host__ __device__ void  CudaMallocAsyncResource::freeBytes(BaseMemoryResource* pAlloc, void* ptr, size_t sz, size_t align) {
#if !defined(__CUDA_ARCH__)
        reinterpret_cast<CudaMallocAsyncResource*>(pAlloc)->deallocate(ptr, sz, align);
#else
        assert(false);
#endif
    }
    __host__ void* CudaMallocAsyncResource::allocateBytesAsync(BaseMemoryResource* pAlloc, size_t sz, size_t align, CudaStreamHandle stream) {
        return reinterpret_cast<CudaMallocAsyncResource*>(pAlloc)->allocate_async(sz, align, streamRefFromHandle(stream));
    }
    __host__ void     CudaMallocAsyncResource::freeBytesAsync(BaseMemoryResource* pAlloc, void* ptr, size_t sz, size_t align, CudaStreamHandle stream) {
        reinterpret_cast<CudaMallocAsyncResource*>(pAlloc)->deallocate_async(ptr, sz, align, streamRefFromHandle(stream));
    }

    __host__ CudaMallocAsyncResource::CudaMallocAsyncResource() : CudaAsyncMemoryReosurce(EMemoryResourceType::eCudaMallocAsync) {
        m_host.allocateBytes = CudaMallocAsyncResource::allocateBytes;
        m_host.freeBytes = CudaMallocAsyncResource::freeBytes;
        m_host.allocateBytesAsync = CudaMallocAsyncResource::allocateBytesAsync;
        m_host.freeBytesAsync = CudaMallocAsyncResource::freeBytesAsync;
        m_host.hostHasAccess = CudaMallocAsyncResource::hostHasAccess;
        m_host.deviceHasAccess = CudaMallocAsyncResource::deviceHasAccess;
        initTable<<<1, 32>>>(*this);
        hipDeviceSynchronize();
    }

    void* CudaMallocAsyncResource::allocate(size_t _Bytes, [[maybe_unused]] size_t _Align)
    {
        void* tmp = nullptr;
        if (hipMalloc(&tmp, _Bytes) != ::hipSuccess)
            return nullptr;
        return tmp;
    }
    void CudaMallocAsyncResource::deallocate(void* _Ptr, size_t _Bytes, [[maybe_unused]] size_t _Align)
    {
        hipError_t err = hipFree(_Ptr);
        assert(err == ::hipSuccess);
    }
    __host__ void* CudaMallocAsyncResource::allocate_async(size_t sz, [[maybe_unused]] size_t align, cuda::stream_ref stream)
    {
        void* tmp = nullptr;
        if (hipMallocAsync(&tmp, sz, stream.get()) != ::hipSuccess)
            return nullptr;
        return tmp;
    }
    __host__ void CudaMallocAsyncResource::deallocate_async(
        void*                   ptr,
        [[maybe_unused]] size_t sz,
        [[maybe_unused]] size_t align,
        cuda::stream_ref        stream)
    {
        hipError_t err = hipFreeAsync(ptr, stream.get());
        assert(err == ::hipSuccess);
    }

    __host__ BuddyMemoryResource::BuddyMemoryResource(BuddyResourceSpec const& input) :
    BaseMemoryResource(makeMemResId(EMemoryResourceType::eHost, EMemoryResourceType::eHostToDevMemMap)),
    m_deviceId(input.deviceId),
    m_minBlockSize(static_cast<uint32_t>(nextPOT(input.minBlockSize)))
    { 
        m_host.allocateBytes = BuddyMemoryResource::allocateBytes;
        m_host.freeBytes = BuddyMemoryResource::freeBytes;
        m_host.allocateBytesAsync = BuddyMemoryResource::allocateBytesAsync;
        m_host.freeBytesAsync = BuddyMemoryResource::freeBytesAsync;
        m_host.deviceHasAccess = BuddyMemoryResource::deviceHasAccess;
        m_host.hostHasAccess = BuddyMemoryResource::hostHasAccess;
        initTable<<<1, 32>>>(*this);
        hipDeviceSynchronize();
        
        // TODO better? allocation functions without class? global context map {pid, idx -> ctx}
        assert(input.pHostMemRes);
        hipError_t res = hipDeviceGet(&m_deviceHnd, m_deviceId);
        assert(res == ::hipSuccess);
        int32_t support = 0;
        res             = hipDeviceGetAttribute(&support, hipDeviceAttributeMemoryPoolsSupported, m_deviceHnd);
        assert(res == ::hipSuccess);
        if (support == 0)
        {
            if (input.pmctx)
                input.pmctx->pctx.error("hipDeviceAttributeMemoryPoolsSupported not supported on the given device");
            std::abort();
        }
        res = hipDeviceGetAttribute(&support, hipDeviceAttributeVirtualMemoryManagementSupported, m_deviceHnd);
        assert(res == ::hipSuccess);
        if (support == 0)
        {
            if (input.pmctx)
                input.pmctx->pctx.error(
                    "hipDeviceAttributeVirtualMemoryManagementSupported not supported on given device");
            std::abort();
        }

        // compute, from max pool size, the amount of virtual address space to reserve on the host for the Control block
        size_t const granularity     = getDeviceAllocationGranularity(m_deviceId);
        m_maxPoolSize                = roundUpToNextMultipleOf(input.maxPoolSize, granularity);
        m_ctrlBlockReservedVMemBytes = sizeof(UnifiedControlBlock) +
                                       ceilDiv(m_maxPoolSize, sizeof(hipMemGenericAllocationHandle_t));
        void* reservedHostSpace = reserveVirtualAddressSpace(m_ctrlBlockReservedVMemBytes);
        if (!reservedHostSpace)
        {
            if (input.pmctx)
                input.pmctx->pctx.error("Couldn't reserve host virtual address space for {} Bytes of metadata",
                                        {m_ctrlBlockReservedVMemBytes});
            std::abort();
        }
        // commit first page, construct control block, compute initial capacity
        if (!commitPhysicalMemory(reservedHostSpace, toUnderlying(EPageSize::e4KB)))
        {
            if (input.pmctx)
                input.pmctx->pctx.error("Couldn't commit first 4KB physical memory page for allocater metadata");
            std::abort();
        }
        size_t initialCapacity = toUnderlying(EPageSize::e4KB) - sizeof(UnifiedControlBlock);
        assert(initialCapacity % sizeof(hipMemGenericAllocationHandle_t) == 0);
        initialCapacity /= sizeof(hipMemGenericAllocationHandle_t);

        m_ctrlBlock = std::construct_at(std::bit_cast<UnifiedControlBlock*>(reservedHostSpace), input.pHostMemRes);
        m_ctrlBlock->refCount.store(1, std::memory_order_seq_cst);
        m_ctrlBlock->capacity = initialCapacity;

        // compute number of initial handles
        m_chunkSize = roundUpToNextMultipleOf(input.minBlocks * static_cast<size_t>(nextPOT(input.minBlockSize)), granularity);
        assert((m_chunkSize < m_maxPoolSize) && (m_maxPoolSize % m_chunkSize == 0));

        // allocate physical memory for first device memory chunk, and reserve virtual address space (has to be paired with `hipMemAddressFree`)
        res = hipMemAddressReserve(&m_ctrlBlock->ptr, m_maxPoolSize, 0 /*natural align*/, 0 /*hint address*/, 0 /*flags*/);
        assert(res == ::hipSuccess);
        hipMemGenericAllocationHandle_t* arr = vlaStart();
        if (!allocateDevicePhysicalMemory(m_deviceId, m_chunkSize, arr[0]))
        {
            if (input.pmctx)
                input.pmctx->pctx.error("Couldn't allocate first Device memory chunk, device memory already exhausted");
            std::abort();
        }
        ++m_ctrlBlock->size;

        // map the first device memory chunk
        res = hipMemMap(m_ctrlBlock->ptr, m_chunkSize, 0, arr[0], 0);
        assert(res == ::hipSuccess);
        setReadWriteDeviceVirtMemory(m_deviceId, m_ctrlBlock->ptr, m_chunkSize);

        // allocate the metadata all in one shot, and memset it to
        assert(m_maxPoolSize % m_minBlockSize == 0);
        size_t const completeBinaryTreeNodes = (m_maxPoolSize / m_minBlockSize) << 1;
        m_ctrlBlock->allocationBitmap.resize(completeBinaryTreeNodes);
        std::ranges::fill(m_ctrlBlock->allocationBitmap, eInvalid);

        // this isn't a "pure" buddy system. The allocation actually has a maximum size, hence subdivide the
        // tree until you get nodes representing blocks of `m_chunkSize`
        size_t const minOrderChunk = minOrder();
        for (size_t level = 0; level < minOrderChunk; ++level)
        {
            size_t numNodes = 1ULL << level;
            for (size_t nodeIdx = 0; nodeIdx < numNodes; ++nodeIdx)
            {
                size_t const index                   = ((1ULL << level) - 1) + nodeIdx; // Calculate index
                m_ctrlBlock->allocationBitmap[index] = eHasChildren;
            }
        }
        for (size_t nodeIdx = 0; nodeIdx < (1ULL << minOrderChunk); ++nodeIdx)
        {
            size_t const index                   = ((1ULL << minOrderChunk) - 1) + nodeIdx; // Calculate index
            m_ctrlBlock->allocationBitmap[index] = eFree;
        }
    }

    __host__ BuddyMemoryResource::BuddyMemoryResource(BuddyMemoryResource const& other) :
    BaseMemoryResource(makeMemResId(EMemoryResourceType::eHost, EMemoryResourceType::eHostToDevMemMap))
    {
        // shouldn't be necessary to lock `transactionInFlight`, cause no device allocation is needed here
        std::shared_lock slk{other.m_mtx};
        m_ctrlBlock                  = other.m_ctrlBlock;
        m_maxPoolSize                = other.m_maxPoolSize;
        m_ctrlBlockReservedVMemBytes = other.m_ctrlBlockReservedVMemBytes;
        m_chunkSize                  = other.m_chunkSize;
        m_deviceHnd                     = other.m_deviceHnd;
        m_deviceId                   = other.m_deviceId;
        m_minBlockSize               = other.m_minBlockSize;

        std::lock_guard lk{m_ctrlBlock->transactionInFlight};
        m_ctrlBlock->refCount.fetch_add(1, std::memory_order_seq_cst);
    }

    __host__ BuddyMemoryResource::BuddyMemoryResource(BuddyMemoryResource&& other) noexcept :
    BaseMemoryResource(makeMemResId(EMemoryResourceType::eHost, EMemoryResourceType::eHostToDevMemMap))
    {
        // shouldn't be necessary to lock `transactionInFlight`, cause no device allocation is needed here
        std::lock_guard lk{other.m_mtx};
        m_ctrlBlock                  = std::exchange(other.m_ctrlBlock, nullptr);
        m_maxPoolSize                = other.m_maxPoolSize;
        m_ctrlBlockReservedVMemBytes = other.m_ctrlBlockReservedVMemBytes;
        m_chunkSize                  = other.m_chunkSize;
        m_deviceHnd                     = other.m_deviceHnd;
        m_deviceId                   = other.m_deviceId;
        m_minBlockSize               = other.m_minBlockSize;
    }

    __host__ BuddyMemoryResource& BuddyMemoryResource::operator=(BuddyMemoryResource const& other)
    {
        if (*this != other)
        {
            cleanup();
            std::unique_lock lk{m_mtx, std::defer_lock};
            std::shared_lock slk{other.m_mtx, std::defer_lock};
            std::lock(lk, slk);

            m_ctrlBlock                  = other.m_ctrlBlock;
            m_maxPoolSize                = other.m_maxPoolSize;
            m_ctrlBlockReservedVMemBytes = other.m_ctrlBlockReservedVMemBytes;
            m_chunkSize                  = other.m_chunkSize;
            m_deviceHnd                     = other.m_deviceHnd;
            m_deviceId                   = other.m_deviceId;
            m_minBlockSize               = other.m_minBlockSize;

            std::lock_guard mlk{m_ctrlBlock->transactionInFlight};
            m_ctrlBlock->refCount.fetch_add(1, std::memory_order_seq_cst);
        }
        return *this;
    }

    __host__ BuddyMemoryResource& BuddyMemoryResource::operator=(BuddyMemoryResource&& other) noexcept
    {
        if (*this != other)
        {
            cleanup();
            std::unique_lock lk{m_mtx, std::defer_lock};
            std::unique_lock olk{other.m_mtx, std::defer_lock};
            std::lock(lk, olk);

            m_ctrlBlock                  = std::exchange(other.m_ctrlBlock, nullptr);
            m_maxPoolSize                = other.m_maxPoolSize;
            m_ctrlBlockReservedVMemBytes = other.m_ctrlBlockReservedVMemBytes;
            m_chunkSize                  = other.m_chunkSize;
            m_deviceHnd                     = other.m_deviceHnd;
            m_deviceId                   = other.m_deviceId;
            m_minBlockSize               = other.m_minBlockSize;
        }
        return *this;
    }

    __host__ void BuddyMemoryResource::cleanup() noexcept
    {
        hipError_t res;
        if (!m_ctrlBlock)
            return;

        { // shared lock scope
            std::shared_lock slk{m_mtx};
            m_ctrlBlock->transactionInFlight.lock();
            if (m_ctrlBlock->refCount.fetch_sub(1, std::memory_order_seq_cst) > 1)
            {
                m_ctrlBlock->transactionInFlight.unlock();
                return;
            }
        }
        std::lock_guard lk{m_mtx};
        if (!m_ctrlBlock)
            return;

        // unmap and deallocate all device memory chunks
        hipMemGenericAllocationHandle_t* arr = vlaStart();
        hipDeviceptr_t                   ptr = m_ctrlBlock->ptr;
        for (size_t i = 0; i < m_ctrlBlock->size; ++i)
        {
            res = hipMemUnmap(ptr, m_chunkSize);
            assert(res == ::hipSuccess);
            res = hipMemRelease(arr[i]);
            assert(res == ::hipSuccess);
            ptr += m_chunkSize;
        }

        // free device virtual address reservation
        res = hipMemAddressFree(m_ctrlBlock->ptr, m_maxPoolSize);
        assert(res == ::hipSuccess);

        std::destroy_at(m_ctrlBlock);

        // compute number of host committed pages, decommit them
        size_t pageCount = m_ctrlBlock->capacity * sizeof(hipMemGenericAllocationHandle_t) + sizeof(UnifiedControlBlock);
        size_t const pageSz = toUnderlying(EPageSize::e4KB);
        assert(pageCount % pageSz == 0);
        pageCount /= pageSz;
        uintptr_t address = std::bit_cast<uintptr_t>(m_ctrlBlock);
        for (size_t i = 0; i < pageCount; ++i)
        { // first iteration will deallocate the lock, so no need to unlock it
            decommitPage(std::bit_cast<void*>(address), pageSz);
            address += pageSz;
        }

        // free host virtual address space reservation
        freeVirtualAddressSpace(std::bit_cast<void*>(m_ctrlBlock), m_ctrlBlockReservedVMemBytes);
        m_ctrlBlock = nullptr;
    }

    hipMemGenericAllocationHandle_t* BuddyMemoryResource::vlaStart() const
    {
        assert(m_ctrlBlock);
        return std::bit_cast<hipMemGenericAllocationHandle_t*>(
            std::bit_cast<uintptr_t>(m_ctrlBlock) + sizeof(UnifiedControlBlock));
    }

    __host__ bool BuddyMemoryResource::grow()
    { // assume you already own the spinlock
        hipError_t res;
        assert(m_ctrlBlock);
        if (m_ctrlBlock->size * m_chunkSize >= m_maxPoolSize)
            return false; // pool exhausted

        // check if capacity is enough
        if (m_ctrlBlock->capacity <= m_ctrlBlock->size)
        {
            void*  address = std::bit_cast<void*>(std::bit_cast<uintptr_t>(m_ctrlBlock) + sizeof(UnifiedControlBlock) +
                                                 sizeof(hipMemGenericAllocationHandle_t) * m_ctrlBlock->capacity);
            size_t pageSz  = static_cast<size_t>(toUnderlying(EPageSize::e4KB));
            assert(pageSz % sizeof(hipMemGenericAllocationHandle_t) == 0);
            if (!commitPhysicalMemory(address, pageSz))
                return false;

            m_ctrlBlock->capacity += pageSz / sizeof(hipMemGenericAllocationHandle_t);
        }

        // allocate and map next chunk of device memory
        size_t const                  offset = m_chunkSize * m_ctrlBlock->size;
        hipMemGenericAllocationHandle_t* arr    = vlaStart();
        if (!allocateDevicePhysicalMemory(m_deviceId, m_chunkSize, arr[m_ctrlBlock->size]))
            return false;

        res = hipMemMap(m_ctrlBlock->ptr + offset, m_chunkSize, 0, arr[m_ctrlBlock->size], 0);
        assert(res == ::hipSuccess);
        setReadWriteDeviceVirtMemory(m_deviceId, m_ctrlBlock->ptr + offset, m_chunkSize);
        ++m_ctrlBlock->size;

        return true;
    }

    __host__ size_t BuddyMemoryResource::minOrder() const
    {
        size_t const lzcntMinSize = std::countl_zero(m_chunkSize);
        size_t const lzcntMaxSize = std::countl_zero(m_maxPoolSize);
        return lzcntMinSize - lzcntMaxSize;
    }

    __host__ size_t BuddyMemoryResource::blockToOrder(size_t size) const
    { // assumptions: size is already a POT between m_minBlockSize and m_chunkSize
        // should return: minOrder()     for size == m_chunkSize
        // should return: minOrder() + n for size == m_chunkSize >> n
        // Ensure the size is a power of two and within valid bounds
        assert(size >= m_minBlockSize && size <= m_chunkSize);
        assert((size & (size - 1)) == 0); // Check if size is a power of two

        // Calculate the minimum order
        size_t const minOrd = minOrder();

        // Calculate the order for the given size
        size_t const order = minOrd + (std::countl_zero(size) - std::countl_zero(m_chunkSize));

        return order;
    }

    __host__ size_t BuddyMemoryResource::alignUpToBlock(size_t size) const
    {
        size_t alignedSize = m_minBlockSize;
        while (alignedSize < size)
        {
            alignedSize <<= 1; // Double the size until it can accommodate 'size'.
        }
        return alignedSize;
    }

    __host__ void BuddyMemoryResource::split(size_t order, size_t nodeIndex)
    { // shared_lock on m_ctx, lock_guard on transactionInFlight acquired by caller
        // Calculate raw array index of the node
        size_t const index = ((1ULL << order) - 1) + nodeIndex;

        // Ensure the node is valid
        assert(index < m_ctrlBlock->allocationBitmap.size());
        assert(m_ctrlBlock->allocationBitmap[index] == eFree || m_ctrlBlock->allocationBitmap[index] == eHasChildren);

        // Mark the current node as having children
        m_ctrlBlock->allocationBitmap[index] = eHasChildren;

        // Calculate the indices of the two children
        size_t const leftChildIndex  = 2 * index + 1;
        size_t const rightChildIndex = 2 * index + 2;
        assert(m_ctrlBlock->allocationBitmap[leftChildIndex] == eInvalid &&
               m_ctrlBlock->allocationBitmap[rightChildIndex] == eInvalid);

        // Ensure indices are within bounds
        assert(leftChildIndex < m_ctrlBlock->allocationBitmap.size());
        assert(rightChildIndex < m_ctrlBlock->allocationBitmap.size());

        // Mark both children as free
        m_ctrlBlock->allocationBitmap[leftChildIndex]  = eFree;
        m_ctrlBlock->allocationBitmap[rightChildIndex] = eFree;
    }

    __host__ bool BuddyMemoryResource::coalesce(size_t parentIndex, size_t parentLevel)
    { // shared_lock on m_ctx, lock_guard on transactionInFlight acquired by caller.
        // parent level > min Order checked by caller
        // Ensure the parent is valid and has children
        assert(parentIndex < m_ctrlBlock->allocationBitmap.size());
        assert(m_ctrlBlock->allocationBitmap[parentIndex] == eHasChildren);

        // Calculate indices of the children
        size_t const leftChildIndex  = 2 * parentIndex + 1;
        size_t const rightChildIndex = 2 * parentIndex + 2;

        // Ensure children indices are within bounds
        assert(leftChildIndex < m_ctrlBlock->allocationBitmap.size());
        assert(rightChildIndex < m_ctrlBlock->allocationBitmap.size());

        // Check if both children are free
        if (m_ctrlBlock->allocationBitmap[leftChildIndex] == eFree && m_ctrlBlock->allocationBitmap[rightChildIndex] == eFree)
        {
            // Mark the parent as free
            m_ctrlBlock->allocationBitmap[parentIndex] = eFree;

            // Clear the children
            m_ctrlBlock->allocationBitmap[leftChildIndex]  = eInvalid;
            m_ctrlBlock->allocationBitmap[rightChildIndex] = eInvalid;

            return true; // Coalescing succeeded
        }

        return false; // Coalescing not possible
    }

    BuddyMemoryResource::~BuddyMemoryResource()
    {
#if defined(__CUDA_ARCH__)
        // you shouldn't be here
#else
        cleanup();
#endif
    }

    __host__ __device__ void* BuddyMemoryResource::allocateBytes(BaseMemoryResource* pAlloc, size_t sz, size_t align)
    {
#if !defined(__CUDA_ARCH__)
            return reinterpret_cast<BuddyMemoryResource*>(pAlloc)->allocate(sz, align);
#else
            return nullptr;
#endif
    }

    __host__ __device__ void BuddyMemoryResource::freeBytes(BaseMemoryResource* pAlloc, void* ptr, size_t sz, size_t align)
    {
#if !defined(__CUDA_ARCH__)
            reinterpret_cast<BuddyMemoryResource*>(pAlloc)->deallocate(ptr, sz, align);
#endif
    }

    __host__ __device__ bool BuddyMemoryResource::deviceHasAccess(BaseMemoryResource const* pAlloc, int32_t deviceID)
    {
        return reinterpret_cast<BuddyMemoryResource const*>(pAlloc)->m_deviceHnd == deviceID;
    }

    __host__ void* BuddyMemoryResource::allocate(size_t _Bytes, size_t _Align)
    {
        if (_Bytes > m_chunkSize)
            return nullptr;

        std::shared_lock slk{m_mtx};
        size_t           alignedSize = alignUpToBlock(_Bytes);
        size_t           level       = blockToOrder(alignedSize);

        std::lock_guard spinGuard{m_ctrlBlock->transactionInFlight};
        // Traverse the allocation bitmap to find a suitable free block
        while (true)
        {
            bool needsGrowth = false;
            for (size_t currentLevel = level; !needsGrowth && currentLevel >= minOrder(); --currentLevel)
            {
                size_t const numNodesAtLevel = 1ULL << currentLevel;

                // Loop through nodes at the current level
                for (size_t nodeIdx = 0; !needsGrowth && nodeIdx < numNodesAtLevel; ++nodeIdx)
                {
                    size_t index = ((1ULL << currentLevel) - 1) + nodeIdx;

                    // Check if the node is free
                    if (m_ctrlBlock->allocationBitmap[index] == eFree)
                    {
                        // Split the block as needed to reach the desired level
                        while (currentLevel < level)
                        {
                            split(currentLevel, nodeIdx); // Split the parent
                            ++currentLevel;
                            nodeIdx *= 2; // Move to the left child
                        }

                        index = ((1ULL << currentLevel) - 1) + nodeIdx;
                        assert(m_ctrlBlock->allocationBitmap[index] == eFree);

                        // Compute the pointer to the allocated memory
                        size_t const offset = nodeIdx * (m_chunkSize >> (currentLevel - minOrder()));
                        hipDeviceptr_t  ptr    = m_ctrlBlock->ptr + offset;
                        hipDeviceptr_t  limit  = m_ctrlBlock->ptr + m_ctrlBlock->size * m_chunkSize;
                        if (ptr < limit)
                        {
                            // Mark the node as allocated
                            m_ctrlBlock->allocationBitmap[index] = eAllocated;
                            return std::bit_cast<void*>(ptr);
                        }
                        else
                        {
                            needsGrowth = true;
                            break;
                        }
                    }
                    else if (m_ctrlBlock->allocationBitmap[index] == eInvalid)
                        break;
                }
            }

            // If no suitable block was found, grow the pool if possible
            if (!grow())
                return nullptr;
        }
    }

    __host__ void BuddyMemoryResource::deallocate(void* _Ptr, size_t _Bytes, size_t _Align)
    {
        std::shared_lock slk{m_mtx};
        // Align the size to the nearest power-of-two block
        _Bytes = alignUpToBlock(_Bytes);

        // Determine the level corresponding to the block size
        size_t const level = blockToOrder(_Bytes);

        // Compute the offset of the block relative to the start of the memory pool
        size_t const offset = std::bit_cast<uintptr_t>(_Ptr) - m_ctrlBlock->ptr;

        // Calculate the index of the block in the allocation bitmap
        size_t const nodeIdx = offset / (m_chunkSize >> (level - minOrder()));
        size_t const index   = ((1ULL << level) - 1) + nodeIdx;

        // Acquire a lock for thread safety
        std::lock_guard lk{m_ctrlBlock->transactionInFlight};

        // Mark the block as free
        assert(m_ctrlBlock->allocationBitmap[index] == eAllocated);
        m_ctrlBlock->allocationBitmap[index] = eFree;

        // Attempt to coalesce blocks up the tree
        size_t parentIndex = (index - 1) / 2;
        size_t parentLevel = level - 1;
        while (parentLevel >= minOrder() && coalesce(parentIndex, parentLevel))
        {
            parentIndex = (parentIndex - 1) / 2; // Move up to the parent
            --parentLevel;
        }
    }

    __host__ __device__ bool BuddyMemoryResource::operator==(BuddyMemoryResource const& that) const noexcept
    { 
        // if control block is the same, then they should be on the same device
        assert((m_ctrlBlock == that.m_ctrlBlock && m_deviceHnd == that.m_deviceHnd) || m_deviceHnd != that.m_deviceHnd);
        return m_deviceHnd == that.m_deviceHnd && m_ctrlBlock == that.m_ctrlBlock;
    }

    // MemPoolAsyncMemoryResource -------------------------------------------------------------------------------------

    __host__ MemPoolAsyncMemoryResource::MemPoolAsyncMemoryResource(MemPoolAsyncMemoryResourceSpec const& input) :
    CudaAsyncMemoryReosurce(EMemoryResourceType::eMemPool),
    m_ctrlBlock(std::bit_cast<ControlBlock*>(input.pHostMemRes->allocate(sizeof(ControlBlock), alignof(ControlBlock)))),
    m_hostCtrlRes(input.pHostMemRes),
    m_poolSize(nextPOT(input.poolSize)),
    m_deviceId(input.deviceId)
    {
        m_host.allocateBytes = MemPoolAsyncMemoryResource::allocateBytes;
        m_host.freeBytes = MemPoolAsyncMemoryResource::freeBytes;
        m_host.allocateBytesAsync = MemPoolAsyncMemoryResource::allocateBytesAsync;
        m_host.freeBytesAsync = MemPoolAsyncMemoryResource::freeBytesAsync;
        m_host.deviceHasAccess = MemPoolAsyncMemoryResource::deviceHasAccess;
        m_host.hostHasAccess = MemPoolAsyncMemoryResource::hostHasAccess;
        initTable<<<1, 32>>>(*this);
        hipDeviceSynchronize();

        // check successful allocation of control block
        hipError_t res;
        if (m_ctrlBlock == nullptr)
        {
            if (input.pmctx)
                input.pmctx->pctx.error("Couldn't allocate control block for async memory resource, aborting...");
            std::abort();
        }
        std::construct_at(m_ctrlBlock);
        m_ctrlBlock->refCount.store(1, std::memory_order_seq_cst);

        // recover device
        hipDevice_t device;
        res = hipDeviceGet(&device, m_deviceId);
        assert(res == ::hipSuccess);

        // query support for used features
        int32_t support = 0;
        res             = hipDeviceGetAttribute(&support, ::hipDeviceAttributeMemoryPoolsSupported, device);
        assert(res == ::hipSuccess);
        if (support == 0)
        {
            if (input.pmctx)
                input.pmctx->pctx.error(
                    "CUDA Memory Pools unsupported, but they are required for this allocator, aborting...");
            std::abort();
        }

        // IPC Support (not needed, here just for reference)
        // Multi-GPU support is instead queried with `hipDeviceCanAccessPeer` and `hipDeviceEnablePeerAccess`
        res = hipDeviceGetAttribute(&support, ::CU_DEVICE_ATTRIBUTE_MEMPOOL_SUPPORTED_HANDLE_TYPES, device);
        assert(res == ::hipSuccess);
        if (support & ::hipMemHandleTypePosixFileDescriptor) // linux (posix) only
            if (input.pmctx)
                input.pmctx->pctx.log(
                    "device {} can create Memory Pools for Inter Process Comunication based on POSIX File Descriptors",
                    {m_deviceId});
        if (support & ::hipMemHandleTypeWin32) // windows only
            if (input.pmctx)
                input.pmctx->pctx
                    .log("device {} can create Memory Pools for Inter Process Communication based on WIN32 Handles",
                         {m_deviceId});
        if (support & ::CU_MEM_HANDLE_TYPE_FABRIC) // CUmemFabricHandle
            if (input.pmctx)
                input.pmctx->pctx.log(
                    "device {} can create Memory Pools for Inter Process Communication based on CUmemFabricHandles",
                    {m_deviceId});
            else if (input.pmctx)
                input.pmctx->pctx.log("device {} cannot export Memory Pools to other processes", {m_deviceId});

        // create the mmeory pool with maximum threashold (meaning the CUDA Runtime won't try to free memory when
        // unoccupied until the pool is destroyed or trimmed explicitly)
        hipMemPoolProps props{};
        props.allocType     = ::hipMemAllocationTypePinned; // only type supported
        props.handleTypes   = ::hipMemHandleTypeNone;       // no IPC
        props.location.type = ::hipMemLocationTypeDevice;
        props.location.id   = m_deviceId;
        props.maxSize       = nextPOT(input.poolSize);
        res                 = hipMemPoolCreate(&m_ctrlBlock->memPool, &props);
        assert(res == ::hipSuccess);

        // set release threshold and reuse policies
        cuuint64_t releaseThreshold = input.releaseThreshold;
        int32_t    enabled          = 1;
        res = hipMemPoolSetAttribute(m_ctrlBlock->memPool, ::hipMemPoolAttrReleaseThreshold, &releaseThreshold);
        assert(res == ::hipSuccess);
        res = hipMemPoolSetAttribute(m_ctrlBlock->memPool, ::hipMemPoolReuseAllowOpportunistic, &enabled);
        assert(res == ::hipSuccess);
        res = hipMemPoolSetAttribute(m_ctrlBlock->memPool, ::hipMemPoolReuseAllowInternalDependencies, &enabled);
        assert(res == ::hipSuccess);
        res = hipMemPoolSetAttribute(m_ctrlBlock->memPool, ::hipMemPoolReuseFollowEventDependencies, &enabled);
        assert(res == ::hipSuccess);

        // create stream for synchronous allocations
        res = hipStreamCreateWithFlags(&m_ctrlBlock->defaultStream, ::hipStreamNonBlocking);
        assert(res == ::hipSuccess);
    }

    __host__ MemPoolAsyncMemoryResource::MemPoolAsyncMemoryResource(MemPoolAsyncMemoryResource const& other) :
    CudaAsyncMemoryReosurce(EMemoryResourceType::eMemPool)
    {
        std::shared_lock slk{other.m_mtx};
        m_ctrlBlock   = other.m_ctrlBlock;
        m_hostCtrlRes = other.m_hostCtrlRes;
        m_deviceId    = other.m_deviceId;
        m_poolSize    = other.m_poolSize;

        std::lock_guard lk{m_ctrlBlock->transactionInFlight};
        m_ctrlBlock->refCount.fetch_add(1, std::memory_order_seq_cst);
    }

    __host__ MemPoolAsyncMemoryResource::MemPoolAsyncMemoryResource(MemPoolAsyncMemoryResource&& other) noexcept :
    CudaAsyncMemoryReosurce(EMemoryResourceType::eMemPool)
    {
        std::lock_guard lk{other.m_mtx};
        m_ctrlBlock   = std::exchange(other.m_ctrlBlock, nullptr);
        m_hostCtrlRes = other.m_hostCtrlRes;
        m_deviceId    = other.m_deviceId;
        m_poolSize    = other.m_poolSize;
    }

    __host__ MemPoolAsyncMemoryResource& MemPoolAsyncMemoryResource::operator=(MemPoolAsyncMemoryResource const& other)
    {
        if (*this != other)
        {
            cleanup();
            std::unique_lock lk{m_mtx, std::defer_lock};
            std::shared_lock slk{other.m_mtx, std::defer_lock};
            std::lock(lk, slk);

            m_ctrlBlock   = other.m_ctrlBlock;
            m_hostCtrlRes = other.m_hostCtrlRes;
            m_deviceId    = other.m_deviceId;
            m_poolSize    = other.m_poolSize;

            std::lock_guard mlk{m_ctrlBlock->transactionInFlight};
            m_ctrlBlock->refCount.fetch_add(1, std::memory_order_seq_cst);
        }
        return *this;
    }

    __host__ MemPoolAsyncMemoryResource& MemPoolAsyncMemoryResource::operator=(MemPoolAsyncMemoryResource&& other) noexcept
    {
        if (*this != other)
        {
            cleanup();
            std::unique_lock lk{m_mtx, std::defer_lock};
            std::unique_lock slk{other.m_mtx, std::defer_lock};
            std::lock(lk, slk);

            m_ctrlBlock   = std::exchange(other.m_ctrlBlock, nullptr);
            m_hostCtrlRes = other.m_hostCtrlRes;
            m_deviceId    = other.m_deviceId;
            m_poolSize    = other.m_poolSize;
        }
        return *this;
    }

    __host__ MemPoolAsyncMemoryResource::~MemPoolAsyncMemoryResource() noexcept
    {
        cleanup();
    }

    __host__ void* MemPoolAsyncMemoryResource::allocate(size_t _Bytes, size_t _Align)
    {
        hipError_t         res;
        std::shared_lock slk{m_mtx};                           // lock to access object
        std::lock_guard  lk{m_ctrlBlock->transactionInFlight}; // lock to allocate memory
        void*            ret = performAlloc(m_ctrlBlock->defaultStream, _Bytes);
        res                  = hipStreamSynchronize(m_ctrlBlock->defaultStream);
        assert(res == ::hipSuccess);
        return ret;
    }

    __host__ void MemPoolAsyncMemoryResource::deallocate(void* _Ptr, size_t _Bytes, size_t _Align)
    {
        hipError_t         res;
        std::shared_lock slk{m_mtx};                           // lock to access object
        std::lock_guard  lk{m_ctrlBlock->transactionInFlight}; // lock to allocate memory
        res = hipFreeAsync(std::bit_cast<hipDeviceptr_t>(_Ptr), m_ctrlBlock->defaultStream);
        assert(res == ::hipSuccess);
        res = hipStreamSynchronize(m_ctrlBlock->defaultStream);
        assert(res == ::hipSuccess);
    }

    __host__ __device__ bool MemPoolAsyncMemoryResource::operator==(MemPoolAsyncMemoryResource const& other) const noexcept
    { // Doesn't work if Multi GPU support is introduced
        bool sameCtrlBlock = other.m_ctrlBlock == m_ctrlBlock;
        bool sameDevice    = m_deviceId == other.m_deviceId;
        assert((sameCtrlBlock && sameDevice) || !sameCtrlBlock);
        return sameDevice && sameCtrlBlock;
    }

    __host__ void* MemPoolAsyncMemoryResource::allocate_async(size_t sz, size_t align, cuda::stream_ref streamRef)
    {
        std::shared_lock slk{m_mtx};                           // lock to access object
        std::lock_guard  lk{m_ctrlBlock->transactionInFlight}; // lock to allocate memory
        hipStream_t         stream = static_cast<hipStream_t>(streamRef.get());
        return performAlloc(stream, sz);
    }

    __host__ void* MemPoolAsyncMemoryResource::performAlloc(hipStream_t stream, size_t sz)
    {
        hipDeviceptr_t ptr = 0;
        hipError_t    res = hipMallocFromPoolAsync(&ptr, sz, m_ctrlBlock->memPool, stream);
        if (res == ::hipSuccess)
            return std::bit_cast<void*>(ptr);
        else if (res == ::hipErrorOutOfMemory)
            return nullptr;
        else
        {
            assert(false);
            return nullptr;
        }
    }

    __host__ __device__ void* MemPoolAsyncMemoryResource::allocateBytes(BaseMemoryResource* pAlloc, size_t sz, size_t align)
    {
#if defined(__CUDA_ARCH__)
        assert(false);
        return nullptr;
#else
        return reinterpret_cast<MemPoolAsyncMemoryResource*>(pAlloc)->allocate(sz, align);
#endif
    }

    __host__ __device__ void MemPoolAsyncMemoryResource::freeBytes(BaseMemoryResource* pAlloc, void* ptr, size_t sz, size_t align)
    {
#if defined(__CUDA_ARCH__)
        assert(false);
#else
        reinterpret_cast<MemPoolAsyncMemoryResource*>(pAlloc)->deallocate(ptr, sz, align);
#endif
    }

    __host__ void* MemPoolAsyncMemoryResource::allocateBytesAsync(BaseMemoryResource* pAlloc, size_t sz, size_t align, CudaStreamHandle stream)
    {
        return reinterpret_cast<MemPoolAsyncMemoryResource*>(pAlloc)->allocate_async(sz, align, streamRefFromHandle(stream));
    }

    __host__ void MemPoolAsyncMemoryResource::freeBytesAsync(BaseMemoryResource* pAlloc, void* ptr, size_t sz, size_t align, CudaStreamHandle stream)
    {
        reinterpret_cast<MemPoolAsyncMemoryResource*>(pAlloc)->deallocate_async(ptr, sz, align, streamRefFromHandle(stream));
    }

    __host__ __device__ bool MemPoolAsyncMemoryResource::deviceHasAccess(BaseMemoryResource const* pAlloc, int32_t deviceID)
    {
		return reinterpret_cast<MemPoolAsyncMemoryResource const*>(pAlloc)->m_deviceId == deviceID;
    }

    __host__ void MemPoolAsyncMemoryResource::deallocate_async(void* ptr, size_t sz, size_t align, cuda::stream_ref streamRef)
    {                                                          // should be called on the same stream as the allocation
        std::shared_lock slk{m_mtx};                           // lock to access object
        std::lock_guard  lk{m_ctrlBlock->transactionInFlight}; // lock to allocate memory
        hipStream_t         stream = static_cast<hipStream_t>(streamRef.get());
        hipError_t         res    = hipFreeAsync(std::bit_cast<hipDeviceptr_t>(ptr), stream);
        assert(res == ::hipSuccess);
    }

    __host__ void MemPoolAsyncMemoryResource::cleanup() noexcept
    {
        hipError_t res;
        if (!m_ctrlBlock)
            return;

        { // shared lock scope (decrement ref counter)
            std::shared_lock slk{m_mtx};
            m_ctrlBlock->transactionInFlight.lock();
            if (m_ctrlBlock->refCount.fetch_sub(1, std::memory_order_seq_cst) > 1)
            {
                m_ctrlBlock->transactionInFlight.unlock();
                return;
            }
        }
        std::lock_guard lk{m_mtx};
        if (!m_ctrlBlock)
            return;

        // destroy memory pool
        hipCtxSynchronize();
        res = hipMemPoolDestroy(m_ctrlBlock->memPool);
        assert(res == ::hipSuccess);

        // destroy default stream
        res = hipStreamDestroy(m_ctrlBlock->defaultStream);
        assert(res == ::hipSuccess);

        // trigger destructor of control block
        std::destroy_at(m_ctrlBlock);
        m_hostCtrlRes->deallocate(m_ctrlBlock, sizeof(ControlBlock), alignof(ControlBlock));
        m_ctrlBlock = nullptr;
    }

    // Memory Resouce Boilerplate -------------------------------------------------------------------------------------
    __host__ __device__ void switchOnMemoryResource(
        EMemoryResourceType eAlloc,
        BaseMemoryResource* p,
        size_t*             sz,
        bool                destroy,
        void*               ctorParam)
    {
        EMemoryResourceType category = extractCategory(eAlloc);
        EMemoryResourceType type     = extractType(eAlloc);
        switch (category)
        {
            using enum EMemoryResourceType;
            case eHost:
                switch (type)
                {
                    case eHostToDevMemMap:
                        if (p)
                            if (destroy)
                                std::destroy_at(std::bit_cast<BuddyMemoryResource*>(p));
                            else
                                std::construct_at(std::bit_cast<BuddyMemoryResource*>(p),
                                                  *std::bit_cast<BuddyResourceSpec*>(ctorParam));
                        else if (sz)
                            *sz = sizeof(BuddyMemoryResource);
                        break;
                }
                break;
            case eDevice:
                switch (type)
                {
                    case eCudaMalloc:
                        if (p)
                            if (destroy)
                                std::destroy_at(std::bit_cast<CudaMallocResource*>(p));
                            else
                                std::construct_at(std::bit_cast<CudaMallocResource*>(p));
                        else if (sz)
                            *sz = sizeof(CudaMallocResource);
                        break;
                }
                break;
            case eAsync:
                switch (type)
                {
                    case eCudaMallocAsync:
                        if (p)
                            if (destroy)
                                std::destroy_at(std::bit_cast<CudaMallocAsyncResource*>(p));
                            else
                                std::construct_at(std::bit_cast<CudaMallocAsyncResource*>(p));
                        else if (sz)
                            *sz = sizeof(CudaMallocAsyncResource);
                        break;
                    case eMemPool:
                        if (p)
                            if (destroy)
                                std::destroy_at(std::bit_cast<MemPoolAsyncMemoryResource*>(p));
                            else
                                std::construct_at(std::bit_cast<MemPoolAsyncMemoryResource*>(p),
                                                  *std::bit_cast<MemPoolAsyncMemoryResourceSpec*>(ctorParam));
                        else if (sz)
                            *sz = sizeof(MemPoolAsyncMemoryResource);
                        break;
                }
                break;
            case eUnified:
                switch (type)
                {
                    case eCudaMallocManaged:
                        if (p)
                            if (destroy)
                                std::destroy_at(std::bit_cast<UnifiedMemoryResource*>(p));
                            else
                                std::construct_at(std::bit_cast<UnifiedMemoryResource*>(p));
                        else if (sz)
                            *sz = sizeof(UnifiedMemoryResource);
                        break;
                }
                break;
        }
    }

    __host__ __device__ size_t sizeForMemoryResource(EMemoryResourceType eAlloc)
    {
        size_t ret = 0;
        switchOnMemoryResource(eAlloc, nullptr, &ret, true, nullptr);
        return ret;
    }

    __host__ BaseMemoryResource* constructMemoryResourceAt(void* ptr, EMemoryResourceType eAlloc, void* ctorParam)
    {
        BaseMemoryResource* p = std::bit_cast<BaseMemoryResource*>(ptr);
        switchOnMemoryResource(eAlloc, p, nullptr, false, ctorParam);
        return p;
    }

    __host__ void destroyMemoryResourceAt(BaseMemoryResource* p, EMemoryResourceType eAlloc)
    {
        switchOnMemoryResource(eAlloc, p, nullptr, true, nullptr);
    }

    __host__ __device__ EMemoryResourceType categoryOf(BaseMemoryResource* allocator)
    {
        return extractCategory(allocator->type);
    }

    __host__ __device__ bool isDeviceAllocator(BaseMemoryResource* allocator, int32_t deviceId)
    {
        return allocator->deviceHasAccess(deviceId);
    }

    __host__ __device__ bool isHostAllocator(BaseMemoryResource* allocator) { return allocator->hostHasAccess(); }

    // BaseDeviceContainer --------------------------------------------------------------------------------------------
    // TODO: use cuda::atomic_ref instead of atomic primitives. Reference: https://nvidia.github.io/cccl/libcudacxx/extended_api/memory_model.html
    __host__ __device__ void BaseDeviceContainer::lockForRead() const
    {
#if defined(__CUDA_ARCH__)
        // Wait until no writer is active
        while (atomicAdd(&m_writeCount, 0) > 0)
        {
            // Spin-wait
        }
        // Increment reader count
        atomicAdd(&m_readCount, 1);
#else
        std::atomic_ref<int> writeRef(m_writeCount);
        std::atomic_ref<int> readRef(m_readCount);
        // Wait until no writer is active
        while (writeRef.load(std::memory_order_acquire) > 0)
        {
            // Spin-wait
        }
        // Increment reader count
        readRef.fetch_add(1, std::memory_order_acquire);
#endif
    }

    __host__ __device__ void BaseDeviceContainer::unlockForRead() const
    {
#if defined(__CUDA_ARCH__)
        // Decrement reader count
        atomicSub(&m_readCount, 1);
#else
        std::atomic_ref<int> readRef(m_readCount);
        // Decrement reader count
        readRef.fetch_sub(1, std::memory_order_release);
#endif
    }

    __host__ __device__ bool BaseDeviceContainer::lockForWrite() const
    {
        int expected = 0;
#if defined(__CUDA_ARCH__)
        // bacause of warp execution (cc < 7.0), the atomic exchange to 1 cannot possibly go right within a warp
        // therefore, *only the "warp leader"* (warpIndex == 0) will lock the thing, and should perform the mutating operations
        // Note: This will cause divergence. once you unlock, you should `__syncthreads` (or `__syncwarp` if cc >= 7.9)
        int32_t warpIndex = warpWideThreadIndex();
        if (warpIndex == leaderWarpIndex)
        {
            while (atomicAdd(&m_readCount, 0) != 0 || atomicCAS(&m_writeCount, expected, 1) != expected)
            {
            }
            return true;
        }
        else
            return false;
#else
        std::atomic_ref<int> writeRef(m_writeCount);
        std::atomic_ref<int> readRef(m_readCount);
        // Wait until no reader or writer is active
        while (readRef.load(std::memory_order_acquire) > 0 ||
               !writeRef.compare_exchange_strong(expected, 1, std::memory_order_seq_cst, std::memory_order_seq_cst))
        {
            // Spin-wait
        }
        return true;
#endif
    }

    __host__ __device__ void BaseDeviceContainer::unlockForWrite() const
    {
#if defined(__CUDA_ARCH__)
        // Decrement writer count
        int32_t warpIndex = warpWideThreadIndex();
        if (warpIndex == leaderWarpIndex)
        {
            atomicExch(&m_writeCount, 0);
        }
#else
        std::atomic_ref<int> writeRef(m_writeCount);
        // Decrement writer count
        writeRef.fetch_sub(1, std::memory_order_release);
#endif
    }

    __host__ __device__ void BaseDeviceContainer::waitWriter() const
    {
#if defined(__CUDA_ARCH__)
        // Spin-wait until no writer is active
        while (atomicAdd(&m_writeCount, 0) > 0)
        {
            // Spin-wait
        }
#else
        std::atomic_ref<int> writeRef(m_writeCount);
        // Spin-wait until no writer is active
        while (writeRef.load(std::memory_order_acquire) > 0)
        {
            // Spin-wait
        }
#endif
    }

    // DynaArray ------------------------------------------------------------------------------------------------------
    __host__ __device__ DynaArray::DynaArray(DynaArray const& other) :
    BaseDeviceContainer(other.m_resource, other.stream),
    m_elemSize(other.m_elemSize)
    {
        other.lockForRead();
        m_resource = other.m_resource;
        stream     = other.stream;
        m_elemSize = other.m_elemSize;
        reserve(other.m_size);
        lockForWrite();
        copyFrom(other);
        unlockForWrite();
        other.unlockForRead();
    }

    __host__ __device__ DynaArray::DynaArray(DynaArray&& other) noexcept :
    BaseDeviceContainer(other.m_resource, other.stream)
    {
        other.lockForWrite();
        m_resource = std::exchange(other.m_resource, nullptr);
        stream     = other.stream;
        m_head     = std::exchange(other.m_head, nullptr);
        m_size     = std::exchange(other.m_size, 0);
        m_capacity = std::exchange(other.m_capacity, 0);
        m_elemSize = other.m_elemSize;
        other.unlockForWrite();
    }

    __host__ __device__ void DynaArray::reserve(size_t newCapacity, bool lock)
    {
        if (lock)
            lockForWrite();
        if (newCapacity <= m_capacity)
        {
            if (lock)
                unlockForWrite();
            return;
        }

        void* newHead = m_resource->tryAllocateAsync(newCapacity * m_elemSize, alignof(std::max_align_t), stream);
        if (m_head)
        {
            std::memcpy(newHead, m_head, m_size * m_elemSize);
            m_resource->tryFreeAsync(m_head, m_size * m_elemSize, alignof(std::max_align_t), stream);
        }
        m_head     = newHead;
        m_capacity = newCapacity;
        if (lock)
            unlockForWrite();
    }

    __host__ __device__ void DynaArray::clear(bool lock) noexcept
    {
        if (lock)
            lockForWrite();
        if (m_head)
        {
            m_resource->tryFreeAsync(m_head, m_size * m_elemSize, alignof(std::max_align_t), stream);
            m_head = nullptr;
        }
        if (lock)
            unlockForWrite();
    }

    __host__ __device__ bool DynaArray::push_back(void const* pValue, bool srcHost, bool lock)
    {
        bool ret = true;
        if (lock)
            lockForWrite();

        if (m_size >= m_capacity)
            reserve(m_capacity > 0 ? m_capacity >> 1 : 1, false);

        void* dest = std::bit_cast<void*>(std::bit_cast<uintptr_t>(m_head) + m_size * m_elemSize);

#if defined(__CUDA_ARCH__)
        if (srcHost) // error!
            ret = false;
        else
            std::memcpy(dest, pValue, m_elemSize);
#else
        int32_t device;
        auto    cudaret = hipGetDevice(&device);
        assert(cudaret == ::hipSuccess);
        hipMemcpyKind kind = isDeviceAllocator(m_resource, device)
                                  ? (srcHost ? ::hipMemcpyHostToDevice : ::hipMemcpyDeviceToDevice)
                                  : (srcHost ? ::hipMemcpyHostToHost : ::hipMemcpyDeviceToHost);
        cudaret             = hipMemcpy(dest, pValue, m_elemSize, kind);
        if (cudaret != ::hipSuccess)
            ret = false;
#endif
        if (ret)
            ++m_size;

        if (lock)
            unlockForWrite();
        return ret;
    }

    __host__ __device__ void DynaArray::pop_back(bool lock)
    {
        if (lock)
            lockForWrite();

        if (m_size != 0)
            --m_size;

        if (lock)
            unlockForWrite();
    }

    // assumes you already locked for read

    __host__ __device__ void* DynaArray::at(size_t index)
    {
        if (eligibleForAccess(index))
            return std::bit_cast<void*>(std::bit_cast<uintptr_t>(m_head) + index * m_elemSize);
        else
            return nullptr;
    }

    __host__ __device__ void const* DynaArray::atConst(size_t index) const
    {
        if (eligibleForAccess(index))
            return std::bit_cast<void const*>(std::bit_cast<uintptr_t>(m_head) + index * m_elemSize);
        else
            return nullptr;
    }

    __host__ __device__ void DynaArray::copyFrom(DynaArray const& other)
    {
        if (other.m_size > 0)
        {
            std::memcpy(m_head, other.m_head, other.m_size * other.m_elemSize);
            m_size = other.m_size;
        }
    }

    __host__ __device__ bool DynaArray::eligibleForAccess(size_t index) const
    {
        assert(index < m_size);
#if defined(__CUDA_ARCH__)
        hipError_t cudaRes;
        int32_t     device;
        cudaRes = hipGetDevice(&device);
        assert(cudaRes == ::hipSuccess);
        if (isDeviceAllocator(m_resource, device))
            return true;
        else
            return false;
#else
        if (isHostAllocator(m_resource))
            return true;
        else
        { // call hipMemcpy? no, wasteful
            assert(false);
            return false;
        }
#endif
    }

    __host__ __device__ DynaArray& DynaArray::operator=(DynaArray const& other)
    {
        if (this != &other)
        {
            lockForWrite();
            other.lockForWrite();
            assert(m_elemSize == other.m_elemSize);

            clear();
            reserve(other.m_size, false);
            copyFrom(other);

            other.unlockForWrite();
            unlockForWrite();
        }
        return *this;
    }

    __host__ __device__ DynaArray& DynaArray::operator=(DynaArray&& other) noexcept
    {
        if (this != &other)
        {
            lockForWrite();
            other.lockForWrite();
            assert(m_elemSize == other.m_elemSize);

            m_size     = std::exchange(other.m_size, 0);
            m_capacity = std::exchange(other.m_capacity, 0);
            m_head     = std::exchange(other.m_head, nullptr);

            other.unlockForWrite();
            unlockForWrite();
        }
        return *this;
    }

    __host__ __device__ DynaArray::~DynaArray() noexcept { clear(); }

    __host__ bool DynaArray::copyToHostSync(void* /*DMT_RESTRICT*/ dest, bool lock) const
    {
        hipError_t err;
        int32_t     device;
        err = hipGetDevice(&device);
        assert(err == ::hipSuccess);
        if (!isDeviceAllocator(m_resource, device))
        {
            assert(false);
            return false;
        }

        bool ret = true;
        if (lock)
            lockForRead();

        ret = ::hipSuccess == hipMemcpy(dest, m_head, m_size * m_elemSize, ::hipMemcpyDeviceToHost);

        if (lock)
            unlockForRead();

        return ret;
    }

    __host__ __device__ size_t DynaArray::size(bool lock) const
    {
        size_t ret = 0;
        if (lock)
            lockForRead();
        ret = m_size;
        if (lock)
            unlockForRead();
        return ret;
    }

    __host__ __device__ size_t DynaArray::capacity(bool lock) const
    {
        size_t ret = 0;
        if (lock)
            lockForRead();
        ret = m_capacity;
        if (lock)
            unlockForRead();
        return ret;
    }
} // namespace dmt
